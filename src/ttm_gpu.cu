#include "hip/hip_runtime.h"
/**
 *
 * OHIO STATE UNIVERSITY SOFTWARE DISTRIBUTION LICENSE
 *
 * Load-balanced sparse MTTKRP on GPUs (the “Software”) Copyright (c) 2019, The Ohio State
 * University. All rights reserved.
 *
 * The Software is available for download and use subject to the terms and
 * conditions of this License. Access or use of the Software constitutes acceptance
 * and agreement to the terms and conditions of this License. Redistribution and
 * use of the Software in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the capitalized paragraph below.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the capitalized paragraph below in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. The names of Ohio State University, or its faculty, staff or students may not
 * be used to endorse or promote products derived from the Software without
 * specific prior written permission.
 *
 * THIS SOFTWARE HAS BEEN APPROVED FOR PUBLIC RELEASE, UNLIMITED DISTRIBUTION. THE
 * SOFTWARE IS PROVIDED “AS IS” AND WITHOUT ANY EXPRESS, IMPLIED OR STATUTORY
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF ACCURACY, COMPLETENESS,
 * NONINFRINGEMENT, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED.  ACCESS OR USE OF THE SOFTWARE IS ENTIRELY AT THE USER’S RISK.  IN
 * NO EVENT SHALL OHIO STATE UNIVERSITY OR ITS FACULTY, STAFF OR STUDENTS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR
 * TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  THE SOFTWARE
 * USER SHALL INDEMNIFY, DEFEND AND HOLD HARMLESS OHIO STATE UNIVERSITY AND ITS
 * FACULTY, STAFF AND STUDENTS FROM ANY AND ALL CLAIMS, ACTIONS, DAMAGES, LOSSES,
 * LIABILITIES, COSTS AND EXPENSES, INCLUDING ATTORNEYS’ FEES AND COURT COSTS,
 * DIRECTLY OR INDIRECTLY ARISING OUT OF OR IN CONNECTION WITH ACCESS OR USE OF THE
 * SOFTWARE.
 *
 */

/**
 *
 * Author:
 *          Israt Nisa (nisa.1@osu.edu)
 *
 * Contacts:
 *          Israt Nisa (nisa.1@osu.edu)
 *          Jiajia Li (jiajia.li@pnnl.gov)
 *
 */

#include <iostream>
#include "ttm_gpu.h"
#include <vector>


int BLOCKSIZE = 512;
dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

inline hipError_t checkCuda(hipError_t result, int s){

  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error in line : %s - %d\n", hipGetErrorString(result), s);
    assert(result == hipSuccess);
  }
  return result;
}

void cuda_timer_start(hipEvent_t start){
    checkCuda(hipEventRecord(start), __LINE__);
}
void cuda_timer_stop(hipEvent_t start, hipEvent_t stop, float &mili){
    checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
}

__global__ void ttm_kernel(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
    ITYPE *fbrPtr1, unsigned int nSlices, DTYPE *dY, DTYPE *dU2, ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

    unsigned int tId = threadIdx.x;
    unsigned int laneId = tId & 31;
    unsigned int gId = (blockIdx.x * blockDim.x + tId);
    unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
    unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
    DTYPE tmp_val;

    if(slc < nSlices){ 
                                      
        unsigned int mappedSlc = dSlcMapperBin[slc];
        unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
        int fb_st = fbrPtr0[mappedSlc];
        int fb_end = fbrPtr0[mappedSlc+1];

        for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
            tmp_val = 0;
            for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

                unsigned int idx2 = dInds2[x];                
                for(unsigned int r=laneId; r<R; r+=32) {
                    tmp_val += vals[x] * dU2[idx2 * R + r]; 
                }
            }
            for(unsigned int r=laneId; r<R; r+=32) {  
                atomicAdd(&dY[fbr * R + r], tmp_val);    
                // dY[fbr * R + r] += tmp_val;  
            }   
        }
    }
}

__global__ void ttm_fbrLevelPar_kernel(DTYPE * vals, ITYPE *dInds2, ITYPE *fbrPtr0,
    ITYPE *fbrPtr1, unsigned int nFibers, DTYPE *dY, DTYPE *dU2, ITYPE   mode, ITYPE R, ITYPE warpPerFiber, int logOfWPF){

    unsigned int tId = threadIdx.x;
    unsigned int laneId = tId & 31;
    unsigned int gId = (blockIdx.x * blockDim.x + tId);
    unsigned int workId = (tId & ((1 << (5 + logOfWPF)) - 1)) >> 5;  
    unsigned int fbr = gId >> (5 + logOfWPF); // 5: minimum 1 WARP (2^5) 
    DTYPE tmp_val;

    if(fbr < nFibers){ 
                                      
        tmp_val = 0;
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerFiber) {

            unsigned int idx2 = dInds2[x];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[x] * dU2[idx2 * R + r]; 
            }
        }
        for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dY[fbr * R + r], tmp_val);    
            // dY[fbr * R + r] += tmp_val;  
        }   
        // }
    }
}

int TTM_GPU(Tensor &X, semiSpTensor &Y, Matrix *U, const Options &Opt){
    //allocate and memcpy GPU memory
    if(Opt.verbose)
        cout << "FIX fiber idx" << endl;
    //Tensor
    ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;
    DTYPE *dVals;
    int logOfWarpPerSlice = log2(Opt.warpPerSlice);
    int TbPerSlc = 1;
    int logOfTPS = log2(TbPerSlc);

    ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

    // dummy bin mapper to be compatible with bin mapper when bin are not used
    X.slcMapperBin.push_back(std::vector<ITYPE>());      
    for (ITYPE s = 0; s < X.fbrIdx[0].size(); ++s)
        X.slcMapperBin[0].push_back(s);

    /* copy tensor metadata */
    checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
    checkCuda(hipMalloc((void**) &dSlcMapperBin, X.slcMapperBin[0].size() * sizeof(ITYPE)), 0);
    checkCuda(hipMalloc((void**) &dfbrIdx0, X.fbrIdx[0].size() * sizeof(ITYPE)), 0);
    checkCuda(hipMalloc((void**) &dfbrPtr0, X.fbrPtr[0].size() * sizeof(ITYPE)), 0);
    checkCuda(hipMalloc((void**) &dfbrPtr1, X.fbrPtr[1].size() * sizeof(ITYPE)), 0);
    checkCuda(hipMalloc((void**) &dfbrIdx1, X.fbrIdx[1].size() * sizeof(ITYPE)), 0);

    checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
    checkCuda(hipMemcpy(dSlcMapperBin, &(X.slcMapperBin[0][0]), X.slcMapperBin[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
    checkCuda(hipMemcpy(dfbrPtr0, &(X.fbrPtr[0][0]), X.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
    checkCuda(hipMemcpy(dfbrIdx0, &(X.fbrIdx[0][0]), X.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
    checkCuda(hipMemcpy(dfbrPtr1, &(X.fbrPtr[1][0]), X.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
    checkCuda(hipMemcpy(dfbrIdx1, &(X.fbrIdx[1][0]), X.fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

    /* copy matrices */
    DTYPE *dU0, *dU1, *dU2, *dU3;   
    checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);
    checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
    
   /* copy interim tensor Y */
    DTYPE *dY;   
    checkCuda(hipMalloc((void**) &dY, Y.nRows * Y.nCols  * sizeof(DTYPE)), 0);
    hipMemset(dY, 0, Y.nRows * Y.nCols * sizeof(DTYPE));

    if(X.ndims == 3){
        checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);
        checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
        
    }

    if(X.ndims == 4){
        ITYPE mode3 = X.modeOrder[3];
        checkCuda(hipMalloc((void**) &dFbrIdx2, X.fbrIdx[2].size() * sizeof(ITYPE)), 0);
        checkCuda(hipMalloc((void**) &dFbrPtr2, X.fbrPtr[2].size() * sizeof(ITYPE)), 0);
        checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
        checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
        
        checkCuda(hipMemcpy(dFbrPtr2, &(X.fbrPtr[2][0]), X.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
        checkCuda(hipMemcpy(dFbrIdx2, &(X.fbrIdx[2][0]), X.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
        checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
        checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
    }

    // BLOCK and GRID
    int BLOCKSIZE = 512;
    block.x = BLOCKSIZE;

    unsigned int warpPerFiber = 16;
    int logOfWarpPerFiber = log2(warpPerFiber);

    if(Opt.warpPerSlice * 32 > BLOCKSIZE){
        cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
        exit(0);
    }

    if(Opt.impType == 2)
        grid.x = (Opt.warpPerSlice * 32 * X.slcMapperBin[0].size() + BLOCKSIZE - 1) / BLOCKSIZE;
    else if(Opt.impType == 3)
        grid.x = (warpPerFiber * 32 * X.nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;


    checkCuda(hipEventRecord(start), __LINE__);

    if(Opt.impType == 2)
        ttm_kernel<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1,
        X.fbrIdx[0].size(), dY, dU2, Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice); 

    /* no notion of slices, launch as many warps as nFibers */

    else if(Opt.impType == 3)
        ttm_fbrLevelPar_kernel<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dInds2, dfbrPtr0, dfbrPtr1, 
        X.nFibers, dY, dU2, Opt.mode, Opt.R, warpPerFiber, logOfWarpPerFiber); 

    checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
    cout << "TTM GPU - time " << mili << "ms"<< endl;

    // check correctness
    checkCuda(hipMemcpy(&Y.vals[0], dY, Y.nRows * Y.nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
    hipFree(dVals);  hipFree(dY);
    hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
    hipFree(dInds2); hipFree(dInds3); 
    hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
    hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);

    return 0;
}

