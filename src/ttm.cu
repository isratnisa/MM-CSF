/**
 *
 * OHIO STATE UNIVERSITY SOFTWARE DISTRIBUTION LICENSE
 *
 * Load-balanced sparse MTTKRP on GPUs (the “Software”) Copyright (c) 2019, The Ohio State
 * University. All rights reserved.
 *
 * The Software is available for download and use subject to the terms and
 * conditions of this License. Access or use of the Software constitutes acceptance
 * and agreement to the terms and conditions of this License. Redistribution and
 * use of the Software in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the capitalized paragraph below.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the capitalized paragraph below in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. The names of Ohio State University, or its faculty, staff or students may not
 * be used to endorse or promote products derived from the Software without
 * specific prior written permission.
 *
 * THIS SOFTWARE HAS BEEN APPROVED FOR PUBLIC RELEASE, UNLIMITED DISTRIBUTION. THE
 * SOFTWARE IS PROVIDED “AS IS” AND WITHOUT ANY EXPRESS, IMPLIED OR STATUTORY
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF ACCURACY, COMPLETENESS,
 * NONINFRINGEMENT, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED.  ACCESS OR USE OF THE SOFTWARE IS ENTIRELY AT THE USER’S RISK.  IN
 * NO EVENT SHALL OHIO STATE UNIVERSITY OR ITS FACULTY, STAFF OR STUDENTS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR
 * TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  THE SOFTWARE
 * USER SHALL INDEMNIFY, DEFEND AND HOLD HARMLESS OHIO STATE UNIVERSITY AND ITS
 * FACULTY, STAFF AND STUDENTS FROM ANY AND ALL CLAIMS, ACTIONS, DAMAGES, LOSSES,
 * LIABILITIES, COSTS AND EXPENSES, INCLUDING ATTORNEYS’ FEES AND COURT COSTS,
 * DIRECTLY OR INDIRECTLY ARISING OUT OF OR IN CONNECTION WITH ACCESS OR USE OF THE
 * SOFTWARE.
 *
 */

/**
 *
 * Author:
 *          Israt Nisa (nisa.1@osu.edu)
 *
 * Contacts:
 *          Israt Nisa (nisa.1@osu.edu)
 *          Jiajia Li (jiajia.li@pnnl.gov)
 *
 */

#include <fstream>
#include <stdio.h>
#include <algorithm>
#include <iterator>
#include <utility>  
#include <math.h> 
#include <omp.h>
#include <hip/hip_runtime.h>
#include "ttm_cpu.h"
#include "ttm_gpu.h"
#include <bits/stdc++.h>  

using namespace std;

int main(int argc, char* argv[]){ 
 
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    Options Opt = parse_cmd_options(argc, argv);

    Tensor X;
    load_tensor(X, Opt);

    create_HCSR(X, Opt);  

    // check if appropriate file is loaded
    string fileNameEndwith;

    fileNameEndwith = to_string(X.modeOrder[0]) ;//+ to_string(X.modeOrder[1]) + to_string(X.modeOrder[2]);
    std::size_t found = Opt.inFileName.find(fileNameEndwith);
    if (found==std::string::npos){
        cout << "Not the correct file for this mode" << endl;
        exit(0);
    }
      
    Matrix *U = new Matrix[X.ndims]; 
    create_mats(X, U, Opt, false);
    randomize_mats(X, U, Opt);
    zero_mat(X, U, Opt.mode);

    //allocate space fro intermediate tensor Y (Y = X * Un)

    semiSpTensor Y;
    cout << "calling allocation " << endl;
    prepare_Y(X, Y, Opt);

    if(Opt.verbose)
        cout << endl << "Starting TTM..." << endl;  
    
    // print tensors and statistics
    if(Opt.impType == 0){
        double t0 = seconds();
        create_HCSR(X, Opt);
        tensor_stats(X);
        // print_HCSRtensor(X);
    }
    // CPU   
    if(Opt.impType == 1){
        double t0 = seconds();
        // ((X.ndims == 3) ?  TTM_COO_CPU(X, U, Opt) :  TTM_COO_CPU_4D(X, U, Opt));  
        TTM_CPU(X, Y, U, Opt); 
        printf("TTM - COO CPU time: %.3f sec \n", seconds() - t0);
    }

    // GPU  
    else if(Opt.impType == 2 || Opt.impType == 3){
        TTM_GPU(X, Y, U, Opt); 
    }
    // // HYB CPU
    // else if(Opt.impType == 9){
    //     create_HCSR(X, Opt);
    //     HYBTensor HybX(X);
    //     cout << "Creating HYB... " ;
    //     double t0 = seconds();
    //     ((X.ndims == 3) ?  create_HYB(HybX, X, Opt) :  create_HYB_4D(HybX, X, Opt));   
    //     printf("create HYB - time: %.3f sec \n", seconds() - t0);

    //     make_HybBin(HybX, Opt);
    //     // print_HYBtensor(HybX);
        
    //     // ((X.ndims == 3) ?  MTTKRP_HYB_CPU(HybX, U, Opt) :  MTTKRP_HYB_CPU_4D(HybX, U, Opt));   
    //     MTTKRP_HYB_GPU(HybX, U, Opt);
        
    // }
    // // // HYB GPU
    // // else if(Opt.impType == 10){
    // //     // MTTKRP_TILED_HCSR_GPU(TiledX, U, Opt);
    // // }


    // // Tiled versions
    // else if(Opt.impType >= 5 && Opt.impType < 9){
        
    //     TiledTensor TiledX[Opt.nTile];
    //     create_HCSR(X, Opt);
    //     // print_HCSRtensor(X);
    //     int tilingMode = X.modeOrder[X.ndims -1];
        
    //     Opt.tileSize = (X.dims[tilingMode] + Opt.nTile - 1)/Opt.nTile;
    //     if(Opt.nTile > X.dims[tilingMode]){
    //         cout << "Number of tiles ("<< Opt.nTile << ") should be as minimum as K's dimension (" << X.dims[tilingMode]  << "). Exiting."<< endl ;
    //         exit(0);
    //     }

    //     // split X into tiles based on K indices
    //     make_KTiling(X, TiledX, Opt);
        
    //     // create HCSR for each tile
    //     for (int tile = 0; tile < Opt.nTile; ++tile){
    //         create_TiledHCSR(TiledX, Opt, tile);
    //         // print_TiledHCSRtensor(TiledX, tile);
    //     }  

    //     // Split tiles into bins accordin to nnz in slice
    //     for (int tile = 0; tile < Opt.nTile; ++tile){
    //         make_TiledBin(TiledX, Opt, tile);
    //     }

    //     // COO GPU  
    //     if(Opt.impType == 5){
    //         double t0 = seconds();
    //         MTTKRP_TILED_COO_CPU(TiledX, U, Opt); 
    //         printf("TILED COO CPU - time: %.3f sec \n", seconds() - t0);  
    //     }

    //      // HCSR GPU  
    //     else if(Opt.impType == 6){
    //         double t0 = seconds();
    //         ((X.ndims == 3) ? MTTKRP_TILED_HCSR_CPU(TiledX, U, Opt) : MTTKRP_TILED_HCSR_CPU_4D(TiledX, U, Opt)); 
    //         printf("TILED HCSR CPU - time: %.3f sec \n", seconds() - t0); 
    //     }  

    //     //COO GPU 
    //     else if(Opt.impType == 7){
    //         cout << "GPU COO has bugs! " << endl;
    //         MTTKRP_TILED_COO_GPU(TiledX, U, Opt);
    //     }

    //     // HCSR GPU
    //     else if(Opt.impType == 8){
    //         MTTKRP_TILED_HCSR_GPU(TiledX, U, Opt);
    //     }
    // }
    // else // e.g. -1 
    //     cout << "no MTTKRP" << endl;

    if(!Opt.outFileName.empty()){
        cout << "Writing Y to " << Opt.outFileName << endl;
        write_output_ttmY(Y, X.modeOrder[0], Opt.outFileName);
    }

    /** Correctness check **/
    if(Opt.correctness){
        cout << "DO COO...now incorrect with fbr threshold " << endl;

        cout << "correctness with CPU " << endl;
        
        if (Opt.impType == 1) {
            cout << "Already running COO seq on CPU!" << endl; 
            exit(0);
        }
        
        int mode = Opt.mode;
        DTYPE *out = (DTYPE*)malloc(Y.nRows * Y.nCols * sizeof(DTYPE));
        memcpy(out, Y.vals, Y.nRows * Y.nCols * sizeof(DTYPE));    
        memset(Y.vals, 0, Y.nRows * Y.nCols * sizeof(DTYPE));
        // ((X.ndims == 3) ?  TTM_CPU(X, U, Opt) :  MTTKRP_COO_CPU_4D(X, U, Opt));   
        TTM_CPU(X, Y, U, Opt);
        correctness_check(out, Y.vals, Y.nRows, Y.nCols);
    }

    free_all(X, Y, U);
}


