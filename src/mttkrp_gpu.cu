#include "hip/hip_runtime.h"
/**
 *
 * OHIO STATE UNIVERSITY SOFTWARE DISTRIBUTION LICENSE
 *
 * Load-balanced sparse MTTKRP on GPUs (the “Software”) Copyright (c) 2019, The Ohio State
 * University. All rights reserved.
 *
 * The Software is available for download and use subject to the terms and
 * conditions of this License. Access or use of the Software constitutes acceptance
 * and agreement to the terms and conditions of this License. Redistribution and
 * use of the Software in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the capitalized paragraph below.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the capitalized paragraph below in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. The names of Ohio State University, or its faculty, staff or students may not
 * be used to endorse or promote products derived from the Software without
 * specific prior written permission.
 *
 * THIS SOFTWARE HAS BEEN APPROVED FOR PUBLIC RELEASE, UNLIMITED DISTRIBUTION. THE
 * SOFTWARE IS PROVIDED “AS IS” AND WITHOUT ANY EXPRESS, IMPLIED OR STATUTORY
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF ACCURACY, COMPLETENESS,
 * NONINFRINGEMENT, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED.  ACCESS OR USE OF THE SOFTWARE IS ENTIRELY AT THE USER’S RISK.  IN
 * NO EVENT SHALL OHIO STATE UNIVERSITY OR ITS FACULTY, STAFF OR STUDENTS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR
 * TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  THE SOFTWARE
 * USER SHALL INDEMNIFY, DEFEND AND HOLD HARMLESS OHIO STATE UNIVERSITY AND ITS
 * FACULTY, STAFF AND STUDENTS FROM ANY AND ALL CLAIMS, ACTIONS, DAMAGES, LOSSES,
 * LIABILITIES, COSTS AND EXPENSES, INCLUDING ATTORNEYS’ FEES AND COURT COSTS,
 * DIRECTLY OR INDIRECTLY ARISING OUT OF OR IN CONNECTION WITH ACCESS OR USE OF THE
 * SOFTWARE.
 *
 */

/**
 *
 * Author:
 *          Israt Nisa (nisa.1@osu.edu)
 *
 * Contacts:
 *          Israt Nisa (nisa.1@osu.edu)
 *          Jiajia Li (jiajia.li@pnnl.gov)
 *
 */

#include <iostream>
#include "mttkrp_gpu.h"
#include <vector>

inline hipError_t checkCuda(hipError_t result, int s){

  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error in line : %s - %d\n", hipGetErrorString(result), s);
    assert(result == hipSuccess);
  }
  return result;
}

void cuda_timer_start(hipEvent_t start){
	checkCuda(hipEventRecord(start), __LINE__);
}
void cuda_timer_stop(hipEvent_t start, hipEvent_t stop, float &mili){
	checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
}

// CUDA kernel call to do COO MTTKRP 
__global__ void mttkrp_COO_kernel(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2,  ITYPE nnz,
	DTYPE *dU0, DTYPE *dU1, DTYPE *dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
            atomicAdd(&dU0[idx0 * R + r], tmp_val);
        }    
	}
}

// CUDA kernel call to do COO MTTKRP using loop
__global__ void mttkrp_COO_kernel_loop(DTYPE * const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE * const dInds2,  const ITYPE nnz,
	DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		x = (gId + nl * nnz_per_loop) >> 5;
		
		if(x < nnz){
	    
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
	            atomicAdd(&dU0[idx0 * R + r], tmp_val);
	        }  
		}
		__syncthreads();
	}
}
// CUDA kernel call to do COO MTTKRP 4D 
__global__ void mttkrp_COO_kernel_4D(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2, ITYPE *dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE *dU1, DTYPE *dU2,  DTYPE *dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];
        ITYPE idx3 = dInds3[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
            atomicAdd(&dU0[idx0 * R + r], tmp_val);
        }    
	}
}

// CUDA kernel call to do COO MTTKRP 4D using loop
__global__ void mttkrp_COO_kernel_4D_loop(DTYPE *const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE *const dInds2, ITYPE * const dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2,  DTYPE * const dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	
	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }
	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) 
	{
		x = (gId + nl * nnz_per_loop) >> 5;

		if(x < nnz){
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];
	        ITYPE idx3 = dInds3[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
	            atomicAdd(&dU0[idx0 * R + r], tmp_val);
	        }
	    }  
	    __syncthreads();  
	}
}
//no atomics because all 1 in HYB - COO 
__global__ void mttkrp_HYB_COO_kernel(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2,  ITYPE nnz,
	DTYPE *dU0, DTYPE *dU1, DTYPE *dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
            dU0[idx0 * R + r] += tmp_val;
        }    
	}
}

// CUDA kernel call to do COO MTTKRP using loop
__global__ void mttkrp_HYB_COO_kernel_loop(DTYPE * const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE * const dInds2,  const ITYPE nnz,
	DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		x = (gId + nl * nnz_per_loop) >> 5;
		
		if(x < nnz){
	    
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
	            dU0[idx0 * R + r] += tmp_val;
	        }  
		}
		__syncthreads();
	}
}

//no atomics because all 1 in HYB - COO 
__global__ void mttkrp_HYB_COO_kernel_4D(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2, ITYPE *dInds3,
  ITYPE nnz,  DTYPE *dU0, DTYPE *dU1, DTYPE *dU2,  DTYPE *dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];
        ITYPE idx3 = dInds3[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] * dU3[idx3 * R + r];
            dU0[idx0 * R + r] += tmp_val;
        }    
	}
}

// CUDA kernel call to do COO MTTKRP 4D using loop
__global__ void mttkrp_HYB_COO_kernel_4D_loop(DTYPE *const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE *const dInds2, ITYPE * const dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2,  DTYPE * const dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	
	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }
	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) 
	{
		x = (gId + nl * nnz_per_loop) >> 5;

		if(x < nnz){
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];
	        ITYPE idx3 = dInds3[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
	            dU0[idx0 * R + r] += tmp_val;
	        }
	    }  
	    __syncthreads();  
	}
}

__global__ void mttkrp_CSL_kernel(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = slc;//dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc]; 
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];
		tmp_val = 0;
		
		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		}
	}
}

__global__ void mttkrp_CSL_kernel_bin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc]; 
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];
		tmp_val = 0;
		
		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		}
	}
}

// CSL kernel with loop like ParTI
__global__ void mttkrp_CSL_kernel_bin_loop(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nSlices > nnz_per_loop) {
        num_loops_nnz = ((nSlices + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		slc = (gId + nl * nnz_per_loop) >> 5;
		              	              
		if(slc < nSlices){ 	    

			unsigned int mappedSlc = dSlcMapperBin[slc];
			unsigned int idx0 = dfbrIdx0[mappedSlc]; 
	    	int fb_st = fbrPtr0[mappedSlc];
			int fb_end = fbrPtr0[mappedSlc+1];
			tmp_val = 0;
			
			for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
				
			    unsigned int idx1 = dInds1[fbr];
		        unsigned int idx2 = dInds2[fbr];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
	            }   
			}
			for(unsigned int r=laneId; r<R; r+=32) {  
				atomicAdd(&dU0[idx0 * R + r], tmp_val);    
			}
		}
		__syncthreads();  
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_CSL_kernel_hvyBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		tmp_val = 0;
		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		} 
	}
}

// HCSR MTTKRP : 16 WARP = 1 TB per slice
__global__ void mttkrp_HCSR_kernel_16WARP(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = tId >> 5; //(tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = blockIdx.x ;//gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0; 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        // unsigned int idx1 = dInds1[fbrPtr1[fbr]]; 
	        unsigned int idx1 = fbrIdx1[fbr];   
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;   // C matrix  
	        }    
		}

		for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dU0[idx0 * R + r], tmp);
        } 
	}
}
// CUDA kernel call to do HCSR MTTKRP for the first bin 1 WARP per slice
__global__ void mttkrp_HCSR_kernel_COO(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int slc = gId >> 5; // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st; fbr < fb_end; fbr++){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	dU0[idx0 * R + r] += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_smllBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	// unsigned int slcPerTb = 16/warpPerSlice;
	// unsigned int shSlc = slc & slcPerTb;
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}

		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp);       
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_smllBin_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE outbuffer = 0, tmp_val = 0, outbuffer1 = 0;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;

		for (int fbrS = fbrPtr0[mappedSlc]; fbrS < fbrPtr0[mappedSlc+1]; fbrS++){
			
			unsigned int idx1 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			outbuffer1 = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx3 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU3[idx3 * R + r]; 
		        }       
		        for(unsigned int r=laneId; r<R; r+=32)  
		        	outbuffer1 += tmp_val * dU2[idx2 * R + r] ;       
		    }
		    for(unsigned int r=laneId; r<R; r+=32) 
	        	outbuffer += outbuffer1 * dU1[idx1 * R + r] ;    
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], outbuffer);  
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_hvyBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1] ; fbr+=warpPerSlice){
			tmp_val = 0;
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];//dInds1[fbrPtr1[fbr]];    
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	            // // atomicAdd(&dU0[idx0 * R + r], tmp);
	        }    
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dU0[idx0 * R + r], tmp);
        } 
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_hvyBin_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE outbuffer = 0, tmp_val = 0, outbuffer1 = 0;;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbrS = fb_st; fbrS < fb_end && fbrS < fbrPtr0[mappedSlc+1] ; fbrS++){
			unsigned int idx1 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			outbuffer1 = 0;

			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
            
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx3 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU3[idx3 * R + r]; 
		        }
		        for(unsigned int r=laneId; r<R; r+=32)  
		        	outbuffer1 += tmp_val * dU2[idx2 * R + r] ;  
		    }
		    for(unsigned int r=laneId; r<R; r+=32) 
	        	outbuffer += outbuffer1 * dU1[idx1 * R + r] ;     
		}
		for(unsigned int r=laneId; r<R; r+=32) { 
            atomicAdd(&dU0[idx0 * R + r], outbuffer);
        } 
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int fbrPerWarp, int logOfFPW){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbr = (gId >> (5 + logOfWPC)) << logOfFPW; // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		bool diffFiber = false;
		unsigned int idx0;

		for (int fr = 0; fr < fbrPerWarp && (fbr+fr) < (nFibers - 1); ++fr){

			diffFiber = false;
			unsigned int idx1 = fbrIdx1[fbr+fr];// dInds1[fbrPtr1[fbr]];  
			idx0 = fbrLikeSlcInds[fbr+fr];//slc;  
 			tmp_val = 0;
 			
	        for(unsigned int x = fbrPtr1[fbr+fr] + workId; x < fbrPtr1[fbr+fr+1]; x+=warpPerSlice) {

		        unsigned int idx2 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; //2MR   
	            }       
	        }
	        	
        	for(unsigned int r=laneId; r<R; r+=32) { 
        		tmp += tmp_val * dU1[idx1 * R + r] ; //2PR
        	} 
	        
        	if(fbrLikeSlcInds[fbr+fr] != fbrLikeSlcInds[fbr+fr+1]) {

        		diffFiber = true;
	        	for(unsigned int r=laneId; r<R; r+=32) { 
	        		atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        	} 
        		tmp = 0;
        	}
        } 

        if(!diffFiber) {  
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	atomicAdd(&dU0[idx0 * R + r], tmp); 
	        }  
        }  
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar_4D(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds3, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, ITYPE nFibers, DTYPE *dU0, 
	DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int fbrPerWarp, int logOfFPW){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbrS = (gId >> (5 + logOfWPC)) << logOfFPW; // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val, tmp2= 0;
		              	              
	if(fbrS < nFibers - 1){ 	    
		
		tmp_val = 0;
		bool diffFiber = false;
		unsigned int idx0;

		for (int fr = 0; fr < fbrPerWarp && (fbrS+fr) < (nFibers - 1); ++fr){

			diffFiber = false;
			unsigned int idx1 = fbrIdx1[fbrS+fr];// dInds1[fbrPtr1[fbr]];  
			idx0 = fbrLikeSlcInds[fbrS+fr];//slc;  
 			tmp = 0;

			for (int fbr = fbrPtr1[fbrS+fr] + workId; fbr < fbrPtr1[fbrS+fr+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
			 
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; x++) {

			        unsigned int idx3 = dInds3[x];

		            for(unsigned int r=laneId; r<R; r+=32) {
		                tmp_val += vals[x] * dU3[idx3 * R + r]; //2MR   
		            }       
		        }
		        	
	        	for(unsigned int r=laneId; r<R; r+=32) { 
	        		tmp += tmp_val * dU2[idx2 * R + r] ;
	        	} 
	        }
	       	for(unsigned int r=laneId; r<R; r+=32) { 
	       		tmp2 += tmp * dU1[idx1 * R + r] ;
	       	} 

        	if(fbrLikeSlcInds[fbrS+fr] != fbrLikeSlcInds[fbrS+fr+1]) {

        		diffFiber = true;
	        	for(unsigned int r=laneId; r<R; r+=32) { 
	        		atomicAdd(&dU0[idx0 * R + r], tmp2); //2PR
	        	} 
        		tmp2 = 0;
        	}
        }

        if(!diffFiber) {  
	        for(unsigned int r=laneId; r<R; r+=32) 
	        	atomicAdd(&dU0[idx0 * R + r], tmp2); //2PR	         
        }  
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_fbr_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	ITYPE slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_fbr_atomic_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE outbuffer = 0, tmp_val = 0, tmp = 0;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;

		for (int fbrS = fbrPtr0[mappedSlc]; fbrS < fbrPtr0[mappedSlc+1]; fbrS++){
			
			unsigned int idx3 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			tmp = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx0 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx1 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU1[idx1 * R + r]; 
		        }

				for(unsigned int r=laneId; r<R; r+=32) {  
		        	tmp = tmp_val * dU2[idx2 * R + r] * dU3[idx3 * R + r] ;  
					atomicAdd(&dU0[idx0 * R + r], tmp);        
		    	}
			}
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_fbr_atomic_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE outbuffer = 0, tmp_val = 0, tmp = 0;;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbrS = fb_st; fbrS < fb_end && fbrS < fbrPtr0[mappedSlc+1] ; fbrS++){
			
			unsigned int idx3 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			tmp = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx0 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx1 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU1[idx1 * R + r]; 
		        }

				for(unsigned int r=laneId; r<R; r+=32) {  
		        	tmp = tmp_val * dU2[idx2 * R + r] * dU3[idx3 * R + r] ;  
					atomicAdd(&dU0[idx0 * R + r], tmp);        
		    	}
			}
		}
	}
}


// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_fbrS_atomic_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val, tmp2 = 0;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		
		unsigned int idx3 = dfbrIdx0[mappedSlc] ;//slc;

		for (int fbrS = fbrPtr0[mappedSlc]; fbrS < fbrPtr0[mappedSlc+1]; fbrS++){
			
			unsigned int idx0 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			tmp = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx1 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx2 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU2[idx2 * R + r]; 
		        }
		        for(unsigned int r=laneId; r<R; r+=32)  
	        		tmp += tmp_val * dU1[idx1 * R + r]  ;  
	        }
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp2 = tmp * dU3[idx3 * R + r];
	        	atomicAdd(&dU0[idx0 * R + r], tmp2); //2PR
	        } 
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_fbrS_atomic_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val, tmp2 = 0;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx3 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbrS = fb_st; fbrS < fb_end && fbrS < fbrPtr0[mappedSlc+1] ; fbrS++){
			
			unsigned int idx0 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			tmp = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx1 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx2 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU2[idx2 * R + r]; 
		        }
		        for(unsigned int r=laneId; r<R; r+=32)  
	        		tmp += tmp_val * dU1[idx1 * R + r]  ;  
	        }
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp2 = tmp * dU3[idx3 * R + r];
	        	atomicAdd(&dU0[idx0 * R + r], tmp2); //2PR
	        } 
		}
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbr = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
		unsigned int idx2 = fbrLikeSlcInds[fbr];//slc; 
		
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

	        unsigned int idx1 = dInds2[x];                    

            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
            }
        }     
        for(unsigned int r=laneId; r<R; r+=32) { 
        	tmp = tmp_val * dU2[idx2 * R + r] ;
        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR

        }    
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_4D(DTYPE * vals, ITYPE *fbrLikeSlcInds,  ITYPE *dInds3, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, ITYPE nFibers, DTYPE *dU0,
	 DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbrS = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val, tmp2 = 0;
		              	              
	if(fbrS < nFibers - 1){ 	    
		
		tmp = 0;
		unsigned int idx2 = fbrLikeSlcInds[fbrS];//slc;  
		unsigned int idx3 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  

        for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
			unsigned int idx0 = fbrIdx2[fbr];
			tmp_val = 0;
	    
		    for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {
				unsigned int idx1 = dInds3[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) 
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            // if(laneId == 0)
	            // printf("from GPU: (%d %d %d %d) - %f %f %f %f \n", idx0, idx1, idx2, idx3, dU0[idx0 * R] , dU1[idx1 * R], dU2[idx2 * R], dU3[idx3 * R]);
	        }
            for(unsigned int r=laneId; r<R; r+=32)  {
	        	tmp = tmp_val * dU2[idx2 * R + r] * dU3[idx3 * R + r] ;  
	        	atomicAdd(&dU0[idx0 * R + r], tmp);
	        }
        }            
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbrS_atomic_fbrLvlPar_4D(DTYPE * vals, ITYPE *fbrLikeSlcInds,  ITYPE *dInds3, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, ITYPE nFibers, DTYPE *dU0,
	 DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbrS = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val, tmp2 = 0;
		              	              
	if(fbrS < nFibers - 1){ 	    
		
		tmp = 0;
		unsigned int idx0 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
		unsigned int idx3 = fbrLikeSlcInds[fbrS];//slc;  
        
        for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
			unsigned int idx1 = fbrIdx2[fbr];
			tmp_val = 0;
	    
		    for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {
				unsigned int idx2 = dInds3[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) 
	                tmp_val += vals[x] * dU2[idx2 * R + r] ; //2MR
	        }
            for(unsigned int r=laneId; r<R; r+=32)  
	        	tmp += tmp_val * dU1[idx1 * R + r]  ;  
        }     
        for(unsigned int r=laneId; r<R; r+=32) { 
        	tmp2 = tmp * dU3[idx3 * R + r];
        	atomicAdd(&dU0[idx0 * R + r], tmp2); //2PR
        }    
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_loop(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);

	//like PARTI
	//hardcoded for 1 warp per nnz
	size_t num_loops_fbr = 1 * 32;
    size_t const fbr_per_loop = gridDim.x * blockDim.x;
    if(nFibers > fbr_per_loop) {
        num_loops_fbr = ((nFibers + fbr_per_loop - 1) / fbr_per_loop) << 5;
    }

	DTYPE tmp = 0, tmp_val;

	unsigned int fbr;

	for(size_t nl=0; nl<num_loops_fbr; ++nl) {
		
		fbr = (gId + nl * fbr_per_loop) >> 5;
		              	              
		if(fbr < nFibers - 1){ 	    
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
			unsigned int idx2 = fbrLikeSlcInds[fbr];//slc;  
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; x++) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_fbr_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	ITYPE laneId = threadIdx.x & 31;
	ITYPE workId = threadIdx.x >> 5;
	ITYPE slc = blockIdx.x >> logOfTPS;
	ITYPE localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; 
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); 
	        }    
		} 
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_all_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	ITYPE slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	// ITYPE slcPerTb = 16/warpPerSlice;
	// ITYPE shSlc = slc & slcPerTb;
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  

            // for(unsigned int r=laneId; r<R; r+=32) 
            // 	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	            	tmp_val =  vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] ;
	            	atomicAdd(&dU0[idx0 * R + r], tmp_val); //2MR
	                // atomicAdd(&dU0[idx0 * R + r], (tmp_val * vals[x]) ); 
	            }
	        }   	
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_all_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	ITYPE laneId = threadIdx.x & 31;
	ITYPE workId = threadIdx.x >> 5;
	ITYPE slc = blockIdx.x >> logOfTPS;
	ITYPE localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]]; 

			for(unsigned int r=laneId; r<R; r+=32) 
            	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ;  
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	            	// atomicAdd(&dU0[idx0 * R + r], (tmp_val * vals[x]) ); 
	            	tmp_val =  vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] ;
	                atomicAdd(&dU0[idx0 * R + r], tmp_val); 
	            }
	        }    
		} 
	}
}


// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_all_atomic_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE outbuffer = 0, tmp_val = 0, tmp = 0;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;

		for (int fbrS = fbrPtr0[mappedSlc]; fbrS < fbrPtr0[mappedSlc+1]; fbrS++){
			
			unsigned int idx2 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
	
			for(unsigned int r=laneId; r<R; r+=32) 
           		tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx3 = fbrIdx2[fbr];
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx0 = dInds3[x];

			       	for(unsigned int r=laneId; r<R; r+=32) {
	                	tmp = vals[x] * dU3[idx3 * R + r] * tmp_val;//2MR
	            		atomicAdd(&dU0[idx0 * R + r], tmp);
	            	}
	            }
			}
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_all_atomic_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbrS = fb_st; fbrS < fb_end && fbrS < fbrPtr0[mappedSlc+1] ; fbrS++){
					
			unsigned int idx2 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
	
			for(unsigned int r=laneId; r<R; r+=32) 
           		tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
			
				ITYPE idx3 = fbrIdx2[fbr];
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx0 = dInds3[x];

			       	for(unsigned int r=laneId; r<R; r+=32) {
	                	tmp = vals[x] * dU3[idx3 * R + r] * tmp_val;//2MR
	            		atomicAdd(&dU0[idx0 * R + r], tmp);
	            	}
	            }
			}
		}
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbr = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		unsigned int idx1 = fbrLikeSlcInds[fbr];//slc;  
		unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  

		// if(laneId == 0 && idx1 == 0)
		// 	printf("GPU %d %d %f %f\n", idx1, idx2, dU1[idx1 * R], dU2[idx2 * R] );

		for(unsigned int r=laneId; r<R; r+=32) 
           	tmp = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
        
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

	        unsigned int idx0 = dInds2[x];                    

            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val = vals[x] * tmp;///dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //2MR
                atomicAdd(&dU0[idx0 * R + r], tmp_val);
            }
        }         
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_4D(DTYPE * vals, ITYPE *fbrLikeSlcInds,  ITYPE *dInds3, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, ITYPE nFibers, DTYPE *dU0,
	 DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	ITYPE fbrS = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val = 0;;
		              	              
	if(fbrS < nFibers - 1){ 	    
		
		tmp = 0;
		unsigned int idx1 = fbrLikeSlcInds[fbrS];//slc;  
		unsigned int idx2 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];                

		for(unsigned int r=laneId; r<R; r+=32) 
           	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR

        for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
			ITYPE idx3 = fbrIdx2[fbr];
	    
		    for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {
				unsigned int idx0 = dInds3[x];  

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp = vals[x] * dU3[idx3 * R + r] * tmp_val;//2MR
	            	atomicAdd(&dU0[idx0 * R + r], tmp);
	            }
	        }
        }            
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_loop(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	ITYPE tId = threadIdx.x;
	ITYPE laneId = tId & 31;
	ITYPE bdim = blockDim.x;
	ITYPE gId = (blockIdx.x * bdim + tId);
	ITYPE warpId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //
	ITYPE blockId = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) //blockIdx.x ;//

	//like PARTI
	//hardcoded for 1 warp per nnz
	size_t num_loops_fbr = 1 * 32;
    size_t const fbr_per_loop = gridDim.x * blockDim.x;
    if(nFibers > fbr_per_loop) {
        num_loops_fbr = ((nFibers + fbr_per_loop - 1) / fbr_per_loop) << 5;
    }

	DTYPE tmp = 0, tmp_val;

	unsigned int fbr;

	for(size_t nl=0; nl<num_loops_fbr; ++nl) {
		
		fbr = (gId + nl * fbr_per_loop) >> 5;
		              	              
		if(fbr < nFibers - 1){ 	    
			
			tmp_val = 0;
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
			unsigned int idx1 = fbrLikeSlcInds[fbr];//slc;  

			for(unsigned int r=laneId; r<R; r+=32) 
	           	tmp = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
	        
	        for(unsigned int x = fbrPtr1[fbr] + warpId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val = vals[x] * tmp;///dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //2MR
	                atomicAdd(&dU0[idx0 * R + r], tmp_val);
	            }
	        }    
		}
	}
}


int MTTKRP_COO_GPU(const Tensor &X, Matrix *U, const Options Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE mode = Opt.mode;
	ITYPE R = Opt.R;
	ITYPE *dInds0, *dInds1, *dInds2, *dInds3;
	DTYPE *dVals;

	ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

	checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds0, X.totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds1, X.totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);

	checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds0, &(X.inds[mode0][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds1, &(X.inds[mode1][0]), X.totNnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	if(X.ndims == 4){
		ITYPE mode3 = X.modeOrder[3];
		checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}
	
	// BLOCK and GRID
	int BLOCKSIZE = 128;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;
    bool useLoop = true;
	
	// /* Like PARTI loop */ = 
	if(useLoop)
		grid.x = 32768;
	else 
		grid.x = (32 * X.totNnz + BLOCKSIZE - 1) / BLOCKSIZE;
	
	// CUDA call
	cuda_timer_start(start);

	if(!useLoop){

		if(X.ndims == 3)
			mttkrp_COO_kernel<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, X.totNnz, dU0, dU1, dU2, mode, R); 
		
		else if(X.ndims == 4)
			mttkrp_COO_kernel_4D<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, dInds3, X.totNnz, dU0, dU1, dU2, dU3, mode, R); 
	
	}
	// /* loop like ParTI */
	else{

		if(X.ndims == 3)
			mttkrp_COO_kernel_loop<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, X.totNnz, dU0, dU1, dU2, mode, R ); 
		
		else if(X.ndims == 4)
			mttkrp_COO_kernel_4D_loop<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, dInds3, X.totNnz, dU0, dU1, dU2, dU3, mode, R); 
	
	}
	cuda_timer_stop(start, stop, mili);

	if(useLoop) cout << "Loop on. ";
    cout << "COO GPU using loop - time " << mili << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	// print_output(U, 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dInds0); hipFree(dInds1); hipFree(dInds2); hipFree(dInds3);


	return 0;
}

int MTTKRP_HCSR_GPU(Tensor &X, Matrix *U, const Options &Opt){
	//allocate and memcpy GPU memory
	cout << "FIX fiber idx" << endl;
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;
	DTYPE *dVals;
	int logOfWarpPerSlice = log2(Opt.warpPerSlice);
	int TbPerSlc = 1;
	int logOfTPS = log2(TbPerSlc);

	ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

	// dummy bin mapper to be compatible with bin mapper when bin are not used
	X.slcMapperBin.push_back(std::vector<ITYPE>());      
	for (int s = 0; s < X.fbrIdx[0].size(); ++s)
		X.slcMapperBin[0].push_back(s);

	checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, X.slcMapperBin[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, X.fbrIdx[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, X.fbrPtr[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, X.fbrPtr[1].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, X.fbrIdx[1].size() * sizeof(ITYPE)), 0);

	checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dSlcMapperBin, &(X.slcMapperBin[0][0]), X.slcMapperBin[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrPtr0, &(X.fbrPtr[0][0]), X.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrIdx0, &(X.fbrIdx[0][0]), X.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrPtr1, &(X.fbrPtr[1][0]), X.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrIdx1, &(X.fbrIdx[1][0]), X.fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	if(X.ndims == 3){
		checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	}

	if(X.ndims == 4){
		ITYPE mode3 = X.modeOrder[3];
		checkCuda(hipMalloc((void**) &dFbrIdx2, X.fbrIdx[2].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, X.fbrPtr[2].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		
		checkCuda(hipMemcpy(dFbrPtr2, &(X.fbrPtr[2][0]), X.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrIdx2, &(X.fbrIdx[2][0]), X.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	grid.x = (Opt.warpPerSlice * 32 * X.dims[mode0] + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;

	checkCuda(hipEventRecord(start), __LINE__);

	// mttkrp_HCSR_kernel_COO<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1, dfbrIdx1,
	// 	X.fbrIdx[0].size(), dU0, dU1, dU2,Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
	if(X.ndims == 3)
		mttkrp_HCSR_kernel_smllBin<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1, dfbrIdx1,
		X.fbrIdx[0].size(), dU0, dU1, dU2,Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
	else
		mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds3, dfbrPtr0, dfbrPtr1, dfbrIdx1,
		dFbrPtr2, dFbrIdx2, X.fbrIdx[0].size(), dU0, dU1, dU2, dU3, Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 


	checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
    cout << "HCSR GPU - time " << mili << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);

	return 0;
}

int MTTKRP_TILED_COO_GPU(TiledTensor *TiledX, Matrix *U, const Options Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE mode = Opt.mode;
	ITYPE R = Opt.R;
	ITYPE *dInds0, *dInds1, *dInds2;
	ITYPE dLoc = 0, totNnz = 0;
	DTYPE *dVals;

	// All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];

	for (int tile = 0; tile < Opt.nTile; ++tile)
		totNnz += TiledX[tile].totNnz;

	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds0, totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds1, totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	for (int tile = 0; tile < Opt.nTile; ++tile){
		
		if(tile > 0) 
			dLoc += TiledX[tile-1].totNnz;

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds0 + dLoc, &(TiledX[tile].inds[mode0][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds1 + dLoc, &(TiledX[tile].inds[mode1][0]), TiledX[tile].totNnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[mode2][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	}

	// //Matrices
	DTYPE *dU0, *dU1, *dU2;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	// BLOCK and GRID
	int BLOCKSIZE = 128;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0, GPUTime = 0;

	// CUDA call
	dLoc = 0;
	for (int tile = 0; tile < Opt.nTile; ++tile){
		
		if(tile > 0) 
			dLoc += TiledX[tile-1].totNnz;

		cout << "Tile " << tile << " launched.. "<<endl;
		
		grid.x = (32 * TiledX[tile].totNnz + BLOCKSIZE - 1) / BLOCKSIZE;

		checkCuda(hipEventRecord(start), __LINE__);
		mttkrp_COO_kernel<<<grid, block>>>(dVals + dLoc, dInds0 + dLoc, dInds1 + dLoc, dInds2 + dLoc, TiledX[tile].totNnz, dU0, dU1, dU2,
								mode, R); 
	
		checkCuda(hipEventRecord(stop), __LINE__);
	    hipEventSynchronize(stop);
	    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
	    hipDeviceSynchronize();
	    cout << "Tile: " << tile << " - time " << mili << "ms"<< endl;
	    GPUTime += mili;
	   
	}
	cout << "COO GPU - time " << GPUTime << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2);
	hipFree(dInds0); hipFree(dInds1); hipFree(dInds2);

	return 0;
}

int MTTKRP_B_HCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){
	
	/*choosing kernel type:
	false: B-CSF- IPDPS work, true: parallelism at fiber level, call slc_atomic_fbrlblpar function*/
	bool slcAtomicFbrLvlPar =  false;

	/* Allocate and memcpy GPU memory */
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// // All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];
    ITYPE mode3 =((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int tile = 0; tile < Opt.nTile; ++tile){
		totNnz += TiledX[tile].totNnz;
		totSlcPtr += TiledX[tile].fbrPtr[0].size() ;
		totSlcIdx += TiledX[tile].fbrIdx[0].size() ;
		totFbrPtr += TiledX[tile].fbrPtr[1].size() ;
		totFbrIdx += TiledX[tile].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[tile].ndims == 4) ? TiledX[tile].fbrPtr[2].size() : 0) ;
	}

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float memcpyTime = 0;
    
    cuda_timer_start(start);
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int tile = 0; tile < Opt.nTile; ++tile){	
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); // all tile same
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[tile].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[tile].fbrPtr[0][0]), TiledX[tile].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[tile].fbrIdx[0][0]), TiledX[tile].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[tile].fbrPtr[1][0]), TiledX[tile].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[tile].fbrIdx[1][0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(slcAtomicFbrLvlPar)
			checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[tile].fbrLikeSlcInds[0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	
		if(TiledX[tile].ndims == 3)
			checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[2]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

		if(TiledX[tile].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[tile].fbrPtr[2][0]), TiledX[tile].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[tile].fbrIdx[2][0]), TiledX[tile].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[tile].inds[mode3][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[tile].slcMapperBin[bin][0]), TiledX[tile].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	cuda_timer_stop(start, stop, memcpyTime);
	cout << "Memcopy time " << memcpyTime << endl;

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per size */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/*MTTKRP on Opt.mode*/
	int MTTKRPmode = mode0;//Opt.mode;

	for (int tile = 0; tile < Opt.nTile; ++tile){

		dBinLoc = 0;
		
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		BLOCKSIZE = (( slcAtomicFbrLvlPar == true) ? Opt.TBsize : 512) ;
		dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

		int smallBinEndsAt = 5;
		int slcPerTb = 0;

		// int warpPerFbr = BLOCKSIZE/32;//1;//Opt.warpPerSlice;//4;//;
		// int logOfWarpPerFbr = log2(warpPerFbr);
		// int bin = 0;
		// int fbrPerWarp = 1;//BLOCKSIZE/32; // dont overflow TB
		// int logOfFbrPerWarp = log2(fbrPerWarp);

		int warpPerFbr =Opt.warpPerSlice;//4;//; BLOCKSIZE/32;//1;//
		int logOfWarpPerFbr = log2(warpPerFbr);
		int fbrPerWarp = Opt.fiberPerWarp;//1;//BLOCKSIZE/32; // dont overflow TB
		int logOfFbrPerWarp = log2(fbrPerWarp );	
		
		grid.x = ( warpPerFbr * 32 * ((TiledX[tile].nFibers+fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;

		double t0 = seconds();
		cuda_timer_start(start);
		
		if(slcAtomicFbrLvlPar){

			if(TiledX[0].ndims == 3)
				mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
				dU0, dU1, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
			else
				mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].nFibers, 
				dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
		}
		
		else{

			for (int bin = 0; bin < Opt.nBin ; ++bin){

				if(bin < smallBinEndsAt){
					
					ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

					dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

					grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(TiledX[0].ndims == 3)
						mttkrp_HCSR_kernel_smllBin<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
				
				// Processing heavy bin.. multiple TB per slice
				else{

					dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
					grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
						if(TiledX[0].ndims == 3)
							mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
							dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
						else
							mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
							dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
			}
		}

		cuda_timer_stop(start, stop, mili);
	    CPUtimer += seconds() - t0;
	    GPUTime += mili;

	    if(Opt.verbose){
	    	cout << "Tile: " << tile << " - time: " << mili << "ms";
	    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
	    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
			cout << endl;
		} 
		
	}
	allModeGPUTime += GPUTime;
	cout << "B-CSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int MTTKRP_B_HCSR_GPU_ANYMODE(TiledTensor *TiledX, Matrix *U, const Options &Opt, int mode){
	
	/* Allocate and memcpy GPU memory */
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// // All tile same mode
	ITYPE mode0 = 0;//TiledX[0].modeOrder[0];
    ITYPE mode1 = 1;//TiledX[0].modeOrder[1];
    ITYPE mode2 = 2;//TiledX[0].modeOrder[2];
    ITYPE mode3 = 3;//((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int tile = 0; tile < Opt.nTile; ++tile){
		totNnz += TiledX[tile].totNnz;
		totSlcPtr += TiledX[tile].fbrPtr[0].size() ;
		totSlcIdx += TiledX[tile].fbrIdx[0].size() ;
		totFbrPtr += TiledX[tile].fbrPtr[1].size() ;
		totFbrIdx += TiledX[tile].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[tile].ndims == 4) ? TiledX[tile].fbrPtr[2].size() : 0) ;
	}

	double t0 = seconds();
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int tile = 0; tile < Opt.nTile; ++tile){	
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); // all tile same
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[tile].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[tile].fbrPtr[0][0]), TiledX[tile].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[tile].fbrIdx[0][0]), TiledX[tile].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[tile].fbrPtr[1][0]), TiledX[tile].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[tile].fbrIdx[1][0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(TiledX[tile].ndims == 3)
			checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[2]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

		if(TiledX[tile].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[tile].fbrPtr[2][0]), TiledX[tile].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[tile].fbrIdx[2][0]), TiledX[tile].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[3]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[tile].slcMapperBin[bin][0]), TiledX[tile].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}

	t0 = seconds();

    unsigned int *dULoc =  new unsigned int[TiledX[0].ndims];
    unsigned int *szDU =  new unsigned int[TiledX[0].ndims];
	
	// //Matrices
	DTYPE *dU;// *dU0, *dU1, *dU2, *dU3;	

	ITYPE mtxSize = ((TiledX[0].ndims == 3) ? (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows) * U[mode0].nCols
		: (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows + U[mode3].nRows) * U[mode0].nCols );
	
	checkCuda(hipMalloc((void**) &dU, mtxSize * sizeof(DTYPE)), 0);

	for (int m = 0; m < TiledX[0].ndims; ++m)
		szDU[m] = U[m].nRows * U[m].nCols;

	ITYPE mtxLoc = 0;
	for (int m = 0; m < mode; ++m)
		mtxLoc += szDU[m];

	checkCuda(hipMemcpy(dU + 0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU + szDU[0], &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU + szDU[0] + szDU[1], &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	if(TiledX[0].ndims == 4)
		checkCuda(hipMemcpy(dU + szDU[0] + szDU[1] + szDU[2], &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	hipMemset(dU + mtxLoc, 0,  U[mode].nRows * U[mode0].nCols * sizeof(DTYPE));

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per size */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/*MTTKRP on Opt.mode*/
	int MTTKRPmode = mode;//Opt.mode;
	
	for (int tile = 0; tile < Opt.nTile; ++tile){

		/* matrix order according to mode order*/ 
		for (int mm = 0; mm < TiledX[0].ndims; ++mm){
			
			int curMode = TiledX[tile].modeOrder[mm];
			dULoc[mm] = 0;
			
			for (int q = 0; q < curMode; ++q)
				dULoc[mm] +=  szDU[q % TiledX[0].ndims]; //1 2 3 0
		}

		dBinLoc = 0;
		
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		// BLOCKSIZE = (( slcAtomicFbrLvlPar == true) ? Opt.TBsize : 512) ;
		dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

		int smallBinEndsAt = 5;
		int slcPerTb = 0;

		// int warpPerFbr = BLOCKSIZE/32;//1;//Opt.warpPerSlice;//4;//;
		// int logOfWarpPerFbr = log2(warpPerFbr);
		// int bin = 0;
		// int fbrPerWarp = 1;//BLOCKSIZE/32; // dont overflow TB
		// int logOfFbrPerWarp = log2(fbrPerWarp);

		int warpPerFbr =Opt.warpPerSlice;//4;//; BLOCKSIZE/32;//1;//
		int logOfWarpPerFbr = log2(warpPerFbr);
		int fbrPerWarp = Opt.fiberPerWarp;//1;//BLOCKSIZE/32; // dont overflow TB
		int logOfFbrPerWarp = log2(fbrPerWarp );	

		double t0 = seconds();
		cuda_timer_start(start);
		
		if(mode == TiledX[0].modeOrder[0]){

			for (int bin = 0; bin < Opt.nBin ; ++bin){

				if(bin < smallBinEndsAt){
					
					ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

					dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

					grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(TiledX[0].ndims == 3)
						mttkrp_HCSR_kernel_smllBin<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
				
				// Processing heavy bin.. multiple TB per slice
				else{

					dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
					grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					if(TiledX[0].ndims == 3)
						mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
			}
		}

		else if(TiledX[0].ndims == 4 && TiledX[0].modeOrder[1] == MTTKRPmode && TiledX[0].totNnz){

			for (int bin = 0; bin < Opt.nBin ; ++bin){

				if(bin < smallBinEndsAt){
					
					ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

					dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

					grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					mttkrp_MIHCSR_kernel_smllBin_fbrS_atomic_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
					dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
				
				// Processing heavy bin.. multiple TB per slice
				else{

					dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
					grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;			
					
					mttkrp_MIHCSR_kernel_hvyBin_fbrS_atomic_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
					dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
			}		
		}
		
		else if(mode == TiledX[0].modeOrder[TiledX[0].ndims-2]){
			
			for (int bin = 0; bin < Opt.nBin ; ++bin){

				if(bin < smallBinEndsAt){
					
					ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

					dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

					grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(TiledX[0].ndims == 3)
						mttkrp_MIHCSR_kernel_smllBin_fbr_atomic<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[1], dU + dULoc[2], dU + dULoc[0], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_MIHCSR_kernel_smllBin_fbr_atomic_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
				
				// Processing heavy bin.. multiple TB per slice
				else{

					dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
					grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					if(TiledX[0].ndims == 3)
						mttkrp_MIHCSR_kernel_hvyBin_fbr_atomic<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[1], dU + dULoc[2], dU + dULoc[0], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_MIHCSR_kernel_hvyBin_fbr_atomic_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
			}
		}
		else if(mode == TiledX[0].modeOrder[TiledX[0].ndims-1]){
			
			for (int bin = 0; bin < Opt.nBin ; ++bin){

				if(bin < smallBinEndsAt){
					
					ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

					dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

					grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(TiledX[0].ndims == 3)
						mttkrp_MIHCSR_kernel_smllBin_all_atomic <<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[2], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_MIHCSR_kernel_smllBin_all_atomic_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
				
				// Processing heavy bin.. multiple TB per slice
				else{

					dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
					grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					if(TiledX[0].ndims == 3)
						mttkrp_MIHCSR_kernel_hvyBin_all_atomic<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[2], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_MIHCSR_kernel_hvyBin_all_atomic_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
			}
		}

		cuda_timer_stop(start, stop, mili);
	    CPUtimer += seconds() - t0;
	    GPUTime += mili;

	    // if(Opt.verbose)
	    {
	    	cout << "Tile: " << tile << " - time: " << mili << "ms";

	    	if(TiledX[0].ndims == 3){
		    	cout << " nSlc: " << TiledX[tile].fbrIdx[0].size() << ", nFibers: "
		    	<< TiledX[tile].fbrPtr[1].size() <<", nnz: "  << TiledX[tile].totNnz;
				cout << endl;
			}
			else if(TiledX[0].ndims == 4){
		    	cout << " nSlc: " << TiledX[tile].fbrIdx[0].size() << ", nSFibers: "
		    	<< TiledX[tile].fbrPtr[1].size() << ", nFibers: "
		    	<< TiledX[tile].fbrPtr[2].size() <<", nnz: "  << TiledX[tile].totNnz;
				cout << endl;
			}
		} 	
	}
	allModeGPUTime += GPUTime;
	cout << "ONE-B-CSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);

	// check correctness
	checkCuda(hipMemcpy(&U[mode].vals[0], dU + mtxLoc, U[mode].nRows * U[mode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	hipFree(dVals); 
	hipFree(dU); //hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int MTTKRP_HYB_GPU(const HYBTensor &HybX, Matrix *U, const Options &Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE *dCOOInds0, *dCOOInds1, *dCOOInds2, *dCOOInds3;
	ITYPE *dCSLSlcPtr, *dCSLSlcInds, *dCSLInds1, *dCSLInds2, *dCSLSlcMapperBin;
	ITYPE *dfbrPtr0, *dfbrIdx0, *dInds2, *dInds3, *dfbrPtr1, *dfbrIdx1,  *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;

	DTYPE *dVals, *dCOOVals, *dCSLVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0, dFbrIdxLoc =0, dBinLoc = 0, dCSLBinLoc = 0, dFbrLoc2 =0;
	int warpPerSlice = Opt.warpPerSlice;
	int logOfWarpPerSlice = log2(Opt.warpPerSlice);
	int TbPerSlc = 1;
	int logOfTPS = log2(TbPerSlc);

	// All tile same mode
	ITYPE mode0 = HybX.modeOrder[0];
    ITYPE mode1 = HybX.modeOrder[1];
    ITYPE mode2 = HybX.modeOrder[2];
    ITYPE mode3 =((HybX.ndims == 4) ? HybX.modeOrder[3] : 0) ;

    // ****** mem op HYB COO *******
    if(HybX.COOnnz > 0){
		
		checkCuda(hipMalloc((void**) &dCOOVals, HybX.COOnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds0, HybX.COOnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds1, HybX.COOnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds2, HybX.COOnnz * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dCOOVals, &(HybX.COOvals[0]), HybX.COOnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds0, &(HybX.COOinds[mode0][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds1, &(HybX.COOinds[mode1][0]), HybX.COOnnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds2, &(HybX.COOinds[mode2][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(HybX.ndims == 4){
			checkCuda(hipMalloc((void**) &dCOOInds3, HybX.COOnnz * sizeof(ITYPE)), 0);
			checkCuda(hipMemcpy(dCOOInds3, &(HybX.COOinds[mode3][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}

   // ****** mem op HYB CSL *******

	if(HybX.CSLnnz > 0){

		checkCuda(hipMalloc((void**) &dCSLVals, HybX.CSLnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcPtr,  HybX.CSLslicePtr.size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcInds, HybX.CSLsliceIdx.size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLInds1, HybX.CSLnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLInds2, HybX.CSLnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcMapperBin, HybX.CSLslicePtr.size() * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dCSLVals, &(HybX.CSLvals[0]), HybX.CSLnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);	
		checkCuda(hipMemcpy(dCSLSlcPtr + dSlcLoc, &(HybX.CSLslicePtr[0]), HybX.CSLslicePtr.size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLSlcInds + dSlcIdxLoc, &(HybX.CSLsliceIdx[0]), HybX.CSLsliceIdx.size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLInds1, &(HybX.CSLinds[mode1][0]), HybX.CSLnnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLInds2, &(HybX.CSLinds[mode2][0]), HybX.CSLnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		dCSLBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dCSLBinLoc += HybX.CSLslcMapperBin[bin-1].size();

			if(HybX.CSLslcMapperBin[bin].size() > 0)
		    	checkCuda(hipMemcpy(dCSLSlcMapperBin + dSlcIdxLoc + dCSLBinLoc, &(HybX.CSLslcMapperBin[bin][0]), HybX.CSLslcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
 
    // ****** mem op HYB HCSR *******

    if(HybX.HCSRnnz > 0){

		checkCuda(hipMalloc((void**) &dVals, HybX.HCSRnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrPtr0,  HybX.fbrPtr[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrIdx0, HybX.fbrIdx[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dSlcMapperBin, HybX.fbrPtr[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrPtr1, HybX.fbrPtr[1].size()  * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrIdx1, HybX.fbrPtr[1].size() * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dVals, &(HybX.vals[0]), HybX.HCSRnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0, &(HybX.fbrPtr[0][0]), HybX.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0, &(HybX.fbrIdx[0][0]), HybX.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1, &(HybX.fbrPtr[1][0]), HybX.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1, &(HybX.fbrIdx[1][0]), HybX.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	    if(HybX.ndims == 3){
	    	checkCuda(hipMalloc((void**) &dInds2, HybX.HCSRnnz * sizeof(ITYPE)), 0);
	    	checkCuda(hipMemcpy(dInds2, &(HybX.inds[mode2][0]), HybX.HCSRnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

	    if(HybX.ndims == 4){
	        checkCuda(hipMalloc((void**) &dFbrIdx2, HybX.fbrIdx[2].size() * sizeof(ITYPE)), 0);
	        checkCuda(hipMalloc((void**) &dFbrPtr2, HybX.fbrPtr[2].size() * sizeof(ITYPE)), 0);
	        checkCuda(hipMalloc((void**) &dInds3, HybX.HCSRnnz * sizeof(ITYPE)), 0);
	        checkCuda(hipMemcpy(dFbrPtr2, &(HybX.fbrPtr[2][0]), HybX.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2, &(HybX.fbrIdx[2][0]), HybX.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3, &(HybX.inds[mode3][0]), HybX.HCSRnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	    }

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += HybX.slcMapperBin[bin-1].size();

			if(HybX.slcMapperBin[bin].size() > 0)
		    	checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(HybX.slcMapperBin[bin][0]), HybX.slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	
	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	if(HybX.ndims == 4){
        checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
        checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
    }
	
	// BLOCK and GRID
	int BLOCKSIZE = 512;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop, HYBstart, HYBstop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&HYBstart);
    hipEventCreate(&HYBstop);

    hipStream_t streams[2 * Opt.nBin + 1];
	for (int bin = 0; bin < 2 * Opt.nBin + 1; ++bin)
		hipStreamCreate(&streams[bin]);

    float mili = 0, HYBmili =0, GPUTime = 0, CPUtimer = 0, HYBTime = 0;
	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0;
	bool useLoop = false;
	
	if(useLoop)
		grid.x = 32768*2;

			// mili = 0; 
	dCSLBinLoc = 0; dBinLoc = 0;

	int smallBinEndsAt = 5;
	int slcPerTb = 0;

	cuda_timer_start(HYBstart);

	// ******* CUDA COO *******

	// if(HybX.COOnnz > 0){

	// 	BLOCKSIZE = 128;
	// 	block.x = BLOCKSIZE;
	// 		// /* Like PARTI loop */ = 

	// 	if(!useLoop)
	// 		grid.x = (32 * HybX.COOnnz + BLOCKSIZE - 1) / BLOCKSIZE;

	// 	if(Opt.verbose) 
	// 		cuda_timer_start(start);
  		
 //  		if(!useLoop){

	//   		if(HybX.ndims == 3)
	// 			mttkrp_HYB_COO_kernel<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
	// 		else if (HybX.ndims == 4)
	// 			mttkrp_HYB_COO_kernel_4D<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
	// 	}

	// 	else{
  			
	//   		if(HybX.ndims == 3)
	// 			mttkrp_HYB_COO_kernel_loop<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
	// 		else if (HybX.ndims == 4)
	// 			mttkrp_HYB_COO_kernel_4D_loop<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
	// 	}

	//     if(Opt.verbose){
	//     	cuda_timer_stop(start, stop, mili);
	//     	HYBTime += mili;
	//     	cout << "HYB-COO GPU " << mili << "ms"<< endl;
	//     }
	// }
	// ******* CUDA CSL *******

	// if(HybX.CSLnnz > 0 || HybX.HCSRnnz > 0)
	{
		if(HybX.COOnnz > 0){

			BLOCKSIZE = 128;
			block.x = 128;
			grid.x = (32 * HybX.COOnnz + BLOCKSIZE - 1) / BLOCKSIZE;

	  		if(HybX.ndims == 3)
				mttkrp_HYB_COO_kernel<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
			else if (HybX.ndims == 4)
				mttkrp_HYB_COO_kernel_4D<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
		
		}

		BLOCKSIZE = 512;
		block.x = BLOCKSIZE;

		for (int bin = 0; bin < Opt.nBin ; ++bin){

			dBinLoc += ((bin > 0) ? HybX.slcMapperBin[bin-1].size() : 0);
			dCSLBinLoc += ((bin > 0) ? HybX.CSLslcMapperBin[bin-1].size() : 0);

			if( HybX.slcMapperBin[bin].size() == 0 && HybX.CSLslcMapperBin[bin].size() == 0)
				continue;
			// Processing small bin.. merged to one. 1 WARP slice
			if(bin < smallBinEndsAt){

				warpPerSlice = 1;
				logOfWarpPerSlice = 0;//log2(warpPerSlice);
				slcPerTb = 16 / warpPerSlice;

				/* CSL small bin */
				if(HybX.CSLnnz > 0){

					grid.x = ( warpPerSlice * 32 * HybX.CSLslcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					mttkrp_CSL_kernel_bin<<<grid, block, 0, streams[1]>>>(dCSLVals, dCSLSlcInds, dCSLSlcMapperBin + dCSLBinLoc, 
						dCSLInds2, dCSLSlcPtr, dCSLInds1, HybX.CSLslcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice); 
				}
				
				/* HCSR small bin */
				if(HybX.HCSRnnz > 0){

					grid.x = ( warpPerSlice * 32 * HybX.slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(HybX.ndims == 3)
						mttkrp_HCSR_kernel_smllBin<<<grid, block, 0, streams[2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, HybX.slcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
					
					else if(HybX.ndims == 4)
						mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, 0, streams[2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, HybX.slcMapperBin[bin].size(), 
						dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
				}
			}

			// Processing heavy bin.. multiple TB per slice
			else{
		
				TbPerSlc = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5

				if(TbPerSlc > 32) TbPerSlc = 32;		
				logOfTPS = log2(TbPerSlc);

				warpPerSlice = 16;
				logOfWarpPerSlice = 4;

				/* CSL big bin */
				if(HybX.CSLnnz > 0){	
					grid.x = (TbPerSlc * warpPerSlice * 32 * HybX.CSLslcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					mttkrp_CSL_kernel_hvyBin<<<grid, block, 0, streams[bin+1]>>>(dCSLVals + dLoc, dCSLSlcInds + dSlcIdxLoc, dCSLSlcMapperBin + dSlcIdxLoc + dCSLBinLoc, 
						dCSLInds2 + dLoc, dCSLSlcPtr + dSlcLoc, dCSLInds1, HybX.CSLslcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS); 
				}

				/* HCSR big bin */
				if(HybX.HCSRnnz > 0){
					grid.x = (TbPerSlc * warpPerSlice * 32 * HybX.slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					if(HybX.ndims == 3)
						mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin+2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, HybX.slcMapperBin[bin].size(), 
							dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS); 
						
					else if(HybX.ndims == 4)
	                    mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin + 2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
	                    dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, HybX.slcMapperBin[bin].size(), 
	                    dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS);
		        } 	

			}
		}

	    // if(Opt.verbose){
	    // 	cuda_timer_stop(start, stop, mili);
	    // 	HYBTime += mili;
	    // 	cout << "CSL+HCSR GPU-time: " << mili << "ms"<< endl;
	    // }
	}

	cuda_timer_stop(HYBstart, HYBstop, HYBmili);
	if(Opt.verbose)
		cout << "verbose on. HYB GPU: " << HYBmili << endl;
	else
		cout << "HYB GPU: " << HYBmili << endl;

	for (int bin = 0; bin < 2 * Opt.nBin + 1; ++bin)
		hipStreamDestroy(streams[bin]);
	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); hipFree(dCOOVals); hipFree(dCSLVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
    hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
    hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dCSLInds1); hipFree(dCSLInds2); hipFree(dCSLSlcPtr); hipFree(dCSLSlcInds);
	hipFree(dCOOInds0); hipFree(dCOOInds1); hipFree(dCOOInds2); 

	return 0;
}

int MTTKRP_ONE_HCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){
	
	bool performMTTKRPMode = true, performMTTKRPnMode = true, performMTTKRPnnMode = true;
	
	/* Allocate and memcpy GPU memory */
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0, dFbrLikeSlcIndsLoc = 0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// // All tile same mode
	ITYPE mode0 = 0;//TiledX[0].modeOrder[0];
    ITYPE mode1 = 1;//TiledX[0].modeOrder[1];
    ITYPE mode2 = 2;//TiledX[0].modeOrder[2];
    ITYPE mode3 = 3;//((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;
    ITYPE R = Opt.R;


	for (int tile = 0; tile < Opt.nTile; ++tile){
		totNnz += TiledX[tile].totNnz;
		totSlcPtr += TiledX[tile].fbrPtr[0].size() ;
		totSlcIdx += TiledX[tile].fbrIdx[0].size() ;
		totFbrPtr += TiledX[tile].fbrPtr[1].size() ;
		totFbrIdx += TiledX[tile].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[tile].ndims == 4) ? TiledX[tile].fbrPtr[2].size() : 0) ;
	}

	double t0 = seconds();
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int tile = 0; tile < Opt.nTile; ++tile){	
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); // all tile same
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[tile].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[tile].fbrPtr[0][0]), TiledX[tile].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[tile].fbrIdx[0][0]), TiledX[tile].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[tile].fbrPtr[1][0]), TiledX[tile].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[tile].fbrIdx[1][0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[tile].fbrLikeSlcInds[0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	
		if(TiledX[tile].ndims == 3)
			checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[2]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

		if(TiledX[tile].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[tile].fbrPtr[2][0]), TiledX[tile].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[tile].fbrIdx[2][0]), TiledX[tile].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[tile].inds[TiledX[0].modeOrder[3]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[tile].slcMapperBin[bin][0]), TiledX[tile].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	float tnsMemcpyTime = seconds() - t0;

	t0 = seconds();

    unsigned int *dULoc =  new unsigned int[TiledX[0].ndims];
    unsigned int *szDU =  new unsigned int[TiledX[0].ndims];
	
	// //Matrices
	DTYPE *dU;// *dU0, *dU1, *dU2, *dU3;	

	ITYPE mtxSize = ((TiledX[0].ndims == 3) ? (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows) * U[mode0].nCols
		: (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows + U[mode3].nRows) * U[mode0].nCols );
	
	checkCuda(hipMalloc((void**) &dU, mtxSize * sizeof(DTYPE)), 0);

	for (int m = 0; m < TiledX[0].ndims; ++m)
		szDU[m] = U[m].nRows * U[m].nCols;

	hipMemset(dU+0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU + szDU[0], &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU + szDU[0] + szDU[1], &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	float mtxMemcpyTime = seconds() - t0;

	// cout << "tns and mtx memcopy time: " << tnsMemcpyTime <<", " << mtxMemcpyTime<< endl;
	
	if(TiledX[0].ndims == 4)
		checkCuda(hipMemcpy(dU + szDU[0] + szDU[1] + szDU[2], &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	// if(Opt.warpPerSlice * 32 > BLOCKSIZE){
	// 	cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
	// 	exit(0);
	// }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per size */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0, dFbrLikeSlcIndsLoc = 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/*MTTKRP on Opt.mode*/

	unsigned int dU0Loc, dU1Loc, dU2Loc , dU3Loc;

	/* matrix order according to mode order*/ 
	for (int m = 0; m < TiledX[0].ndims; ++m){
		
		int curMode = TiledX[0].modeOrder[m];
		dULoc[m] = 0;

		for (int q = 0; q < curMode; ++q){
			dULoc[m] +=  szDU[q % TiledX[0].ndims]; //1 2 3 0
		}
	}
	
	for (int MTTKRPmode = 0; MTTKRPmode < TiledX[0].ndims; ++MTTKRPmode){

		if(MTTKRPmode > 0){

			mili = 0; GPUTime = 0; CPUtimer = 0;
			dLoc = 0; dSlcLoc = 0; dSlcIdxLoc = 0; dFbrLoc =0; dFbrIdxLoc = 0; dFbrLoc2= 0, dFbrLikeSlcIndsLoc = 0;

			// MTTKRP on mode mode 0 changed DU0. To pass correctness for now initializing to 2 again.
			int mode = MTTKRPmode - 1;
		    for(long r = 0; r < U[mode].nRows; ++r){
		        for(long c = 0; c < U[mode].nCols; ++c) // or u[mode].nCols 
		            U[mode].vals[r * U[mode].nCols + c] = mode + .5;// 0.1 * drand48(); //1 ;//(r * R + c + 1); //
		    }

		    if(MTTKRPmode == 1){
		    	checkCuda(hipMemcpy(dU + 0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU + szDU[0], 0,  U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE));
			}
			else if(MTTKRPmode == 2){
				checkCuda(hipMemcpy(dU + szDU[0], &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU + szDU[0] + szDU[1], 0,  U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE));
			}
			else if(MTTKRPmode == 3){
				checkCuda(hipMemcpy(dU + szDU[0] + szDU[1] , &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU + szDU[0] + szDU[1] + szDU[2], 0,  U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE));
			}
		}

		if(performMTTKRPMode && TiledX[0].modeOrder[0] == MTTKRPmode){

			// if(Opt.verbose)
				cout << "Slc atomics - " ;
			
			for (int tile = 0; tile < Opt.nTile; ++tile){

				dBinLoc = 0;
				
				if(tile > 0) {
					dLoc += TiledX[tile-1].totNnz;
					dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
				}
				
				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
				int smallBinEndsAt = 5;
				int slcPerTb = 0;
				int warpPerFbr =Opt.warpPerSlice;//4;//; BLOCKSIZE/32;//1;//
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;
				bool useLoop = false;
				int fbrPerWarp = Opt.fiberPerWarp;//1;//BLOCKSIZE/32; // dont overflow TB
				int logOfFbrPerWarp = log2(fbrPerWarp );
				// int fbrPerWarp = 1;//BLOCKSIZE/32; // dont overflow TB
				// int logOfFbrPerWarp = log2(fbrPerWarp );

				if( (warpPerFbr > (BLOCKSIZE/32)) || (fbrPerWarp > (BLOCKSIZE/32)) ){
					cout << "warpPerFbr (-w) or fbrPerWarp (-s) cannot be higher than threadblock size!"
					<< endl << "hint: increase -b!" << endl;
					exit(0);
				}		

				/* Like PARTI loop */ 
				if(useLoop)
					grid.x = Opt.gridSize;// 32768*16; 
				else 
					grid.x = ( warpPerFbr * 32 * ((TiledX[tile].nFibers+fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;
				
				double t0 = seconds();
				cuda_timer_start(start);
				
				if(TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
					dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
		
				else if(TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[tile].nFibers, dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
				
				cuda_timer_stop(start, stop, mili);
			    CPUtimer += seconds() - t0;
			    GPUTime += mili;

			    if(Opt.verbose){
			    	cout << "Tile: " << tile << " - time: " << mili << "ms";
			    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
			    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
					cout << endl;
				} 
			}
			allModeGPUTime += GPUTime;
			cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;
		}

		/*processing fbrS level for 4D tensor*/
		else if(TiledX[0].ndims == 4 && performMTTKRPnMode && TiledX[0].modeOrder[1] == MTTKRPmode){

			// if(Opt.verbose)
				cout << "FbrS atomics - " ;

			mili = 0, GPUTime = 0, CPUtimer = 0;
			dLoc = 0; dSlcLoc = 0; dSlcIdxLoc = 0; dFbrLoc =0; dFbrIdxLoc = 0; dFbrLoc2= 0, dFbrLikeSlcIndsLoc = 0;

			for (int tile = 0; tile < Opt.nTile; ++tile){

				dBinLoc = 0;
				
				if(tile > 0) {
					dLoc += TiledX[tile-1].totNnz;
					dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
				}
				// cout <<"might wanna change binning style and Block size, logWPC, COO like parallelism, allow mode sort" << endl;

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int smallBinEndsAt = 5;
				int slcPerTb = 0;
				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;

				grid.x = ( warpPerFbr * 32 * TiledX[tile].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				double t0 = seconds();
				cuda_timer_start(start);
									
				mttkrp_MIHCSR_kernel_fbrS_atomic_fbrLvlPar_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
				TiledX[tile].nFibers, dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				
				cuda_timer_stop(start, stop, mili);
			    CPUtimer += seconds() - t0;
			    GPUTime += mili;

			    if(Opt.verbose){
			    	cout << "Tile: " << tile << " - time: " << mili << "ms";
			    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
			    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
					cout << endl;
				} 
			}
			allModeGPUTime += GPUTime;
			cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;
		}

		else if(performMTTKRPnMode && TiledX[0].modeOrder[TiledX[0].ndims-2] == MTTKRPmode){

			// if(Opt.verbose)
				cout << "Fbr atomics - " ;

			mili = 0, GPUTime = 0, CPUtimer = 0;
			dLoc = 0; dSlcLoc = 0; dSlcIdxLoc = 0; dFbrLoc =0; dFbrIdxLoc = 0; dFbrLoc2= 0, dFbrLikeSlcIndsLoc = 0;
			
			for (int tile = 0; tile < Opt.nTile; ++tile){

				dBinLoc = 0;
				
				if(tile > 0) {
					dLoc += TiledX[tile-1].totNnz;
					dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
				}
				// cout <<"might wanna change binning style and Block size, logWPC, COO like parallelism, allow mode sort" << endl;

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int smallBinEndsAt = 5;
				int slcPerTb = 0;
				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;
				bool useLoop = false;

				// /* Like PARTI loop */ = 
				if(useLoop)
					grid.x = Opt.gridSize;// 32768*16; 
				else 
					grid.x = ( warpPerFbr * 32 * TiledX[tile].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				double t0 = seconds();
				cuda_timer_start(start);
				
				if(useLoop)
					mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_loop<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
				dU + dULoc[1], dU + dULoc[2], dU + dULoc[0], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				
				else{
					
					if(TiledX[0].ndims == 3)
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
						dU + dULoc[1], dU + dULoc[2], dU + dULoc[0], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
					
					else if (TiledX[0].ndims == 4)
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
						TiledX[tile].nFibers,  dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}

				cuda_timer_stop(start, stop, mili);
			    CPUtimer += seconds() - t0;
			    GPUTime += mili;

			    if(Opt.verbose){
			    	cout << "Tile: " << tile << " - time: " << mili << "ms";
			    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
			    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
					cout << endl;
				} 
			}
			allModeGPUTime += GPUTime;
			cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;
		} 

		else if(performMTTKRPnnMode && TiledX[0].modeOrder[TiledX[0].ndims-1] == MTTKRPmode){

			// if(Opt.verbose)
				cout << "Nnz atomics - " ;

			mili = 0, GPUTime = 0, CPUtimer = 0;
			dLoc = 0; dSlcLoc = 0; dSlcIdxLoc = 0; dFbrLoc =0; dFbrIdxLoc = 0; dFbrLoc2= 0, dFbrLikeSlcIndsLoc = 0;
			
			for (int tile = 0; tile < Opt.nTile; ++tile){

				dBinLoc = 0;
				
				if(tile > 0) {
					dLoc += TiledX[tile-1].totNnz;
					dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
				}

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				bool useLoop = false;
				int smallBinEndsAt = 5;
				int slcPerTb = 0;
				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;
				
				// /* Like PARTI loop */ = 
				if(useLoop)
					grid.x = Opt.gridSize;// 32768;
				else 
					grid.x = ( warpPerFbr * 32 * TiledX[tile].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				int dloc = 0;
				
				double t0 = seconds();
				cuda_timer_start(start);

				if(useLoop)
					mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_loop<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
				dU + dULoc[2], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				
				else{

					if (TiledX[0].ndims == 3)
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
						dU + dULoc[2], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr); 

					else if (TiledX[0].ndims == 4)
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
						TiledX[tile].nFibers,  dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
				
				cuda_timer_stop(start, stop, mili);
			    CPUtimer += seconds() - t0;
			    GPUTime += mili;

			    if(Opt.verbose){
			    	cout << "Tile: " << tile << " - time: " << mili << "ms";
			    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
			    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
					cout << endl;
				} 
			} 
			allModeGPUTime += GPUTime; 
			cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;
		}
	}
	
	cout << "Total GPU time: " << allModeGPUTime << ", nnz:" << TiledX[0].totNnz 
		<< ", nFibers:" << TiledX[0].fbrPtr[1].size() << ", nSlc:" << TiledX[0].fbrIdx[0].size()
		<< endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);
	
	/* Copying output matrix from GPU to CPU for correctness check */
	int MTTKRPmode = TiledX[0].ndims - 1;
	ITYPE loc = ((TiledX[0].ndims == 3) ? szDU[0] + szDU[1] : szDU[0] + szDU[1] + szDU[2]);
	checkCuda(hipMemcpy(&U[MTTKRPmode].vals[0], dU + loc, U[MTTKRPmode].nRows * U[MTTKRPmode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	// check correctness
	// if(Opt.impType == 14){
	// 	MTTKRPmode = 3;
	// 	checkCuda(hipMemcpy(&U[MTTKRPmode].vals[0] , dU + szDU[0] +szDU[1] + szDU[2], U[MTTKRPmode].nRows * U[MTTKRPmode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	// }
	// else
	// 	checkCuda(hipMemcpy(&U[mode0].vals[0], dU, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU); //hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int MTTKRP_MIHCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){

	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float memcpyTime = 0;

	// All m same mode
	ITYPE mode0 = 0;//TiledX[0].modeOrder[0];
    ITYPE mode1 = 1;;//TiledX[0].modeOrder[1];
    ITYPE mode2 = 2;//TiledX[0].modeOrder[2];
    ITYPE mode3 = 3;//((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int m = 0; m < TiledX[0].ndims; ++m){
		
		if (TiledX[m].totNnz == 0) continue;
		
		totNnz += TiledX[m].totNnz;
		totSlcPtr += TiledX[m].fbrPtr[0].size() ;
		totSlcIdx += TiledX[m].fbrIdx[0].size() ;
		totFbrPtr += TiledX[m].fbrPtr[1].size() ;
		totFbrIdx += TiledX[m].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[m].ndims == 4) ? TiledX[m].fbrPtr[2].size() : 0) ;
	}

	//allocate and memcpy GPU memory
	//Tensor
	cuda_timer_start(start);
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	// checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int m = 0; m < TiledX[0].ndims; ++m){	

		if(m > 0) {

			if (TiledX[m-1].totNnz > 0) {
			
				dLoc += TiledX[m-1].totNnz;
				dSlcLoc += TiledX[m - 1].fbrPtr[0].size(); // all m same
				dSlcIdxLoc += TiledX[m - 1].fbrIdx[0].size(); 
				dFbrLoc += TiledX[m - 1].fbrPtr[1].size();
				dFbrIdxLoc += TiledX[m - 1].fbrIdx[1].size();
				dFbrLoc2 += ((TiledX[m].ndims == 4) ? TiledX[m - 1].fbrPtr[2].size() : 0) ;
			}
		}

		if (TiledX[m].totNnz == 0) continue;

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[m].vals[0]), TiledX[m].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[m].fbrPtr[0][0]), TiledX[m].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[m].fbrIdx[0][0]), TiledX[m].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[m].fbrPtr[1][0]), TiledX[m].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[m].fbrIdx[1][0]), TiledX[m].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[m].fbrLikeSlcInds[0]), TiledX[m].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

		if(TiledX[m].ndims == 3){
			if(m == 0)
				// checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[mode2][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
				checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

			else if(m == 1)
				checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
			else if(m == 2)
				checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
		}
		if(TiledX[m].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[m].fbrPtr[2][0]), TiledX[m].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[m].fbrIdx[2][0]), TiledX[m].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[3]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	cuda_timer_stop(start, stop, memcpyTime);
	cout << "CPU to GPU Memcopy time: " << memcpyTime << endl;

	// //Matrices
	unsigned int *dULoc =  new unsigned int[TiledX[0].ndims];
    unsigned int *szDU =  new unsigned int[TiledX[0].ndims];
	
	// //Matrices
	DTYPE *dU;// *dU0, *dU1, *dU2, *dU3;	

	ITYPE mtxSize = ((TiledX[0].ndims == 3) ? (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows) * U[mode0].nCols
		: (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows + U[mode3].nRows) * U[mode0].nCols );
	
	checkCuda(hipMalloc((void**) &dU, mtxSize * sizeof(DTYPE)), 0);

	for (int m = 0; m < TiledX[0].ndims; ++m)
		szDU[m] = U[m].nRows * U[m].nCols;

	hipMemset(dU+0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU + szDU[0], &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU + szDU[0] + szDU[1], &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	if(TiledX[0].ndims == 4)
		checkCuda(hipMemcpy(dU + szDU[0] + szDU[1] + szDU[2], &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	// if(Opt.warpPerSlice * 32 > BLOCKSIZE){
	// 	cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
	// 	exit(0);
	// }

    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per slice */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	for (int MTTKRPmode = 0; MTTKRPmode < TiledX[0].ndims; ++MTTKRPmode){

		if(MTTKRPmode > 0){

			mili = 0; GPUTime = 0; CPUtimer = 0;
			dLoc = 0; dSlcLoc = 0; dSlcIdxLoc = 0; dFbrLoc =0; dFbrIdxLoc = 0; dFbrLoc2= 0;

			// MTTKRP on mode mode 0 changed DU0. To pass correctness for now initializing to 2 again.
			int mode = MTTKRPmode - 1;
		    for(long r = 0; r < U[mode].nRows; ++r){
		        for(long c = 0; c < U[mode].nCols; ++c) // or u[mode].nCols 
		            U[mode].vals[r * U[mode].nCols + c] = mode + .5;// 0.1 * drand48(); //1 ;//(r * R + c + 1); //
		    }

		    if(MTTKRPmode == 1){
		    	checkCuda(hipMemcpy(dU + 0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU + szDU[0], 0,  U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE));
			}
			else if(MTTKRPmode == 2){
				checkCuda(hipMemcpy(dU + szDU[0], &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU + szDU[0] + szDU[1], 0,  U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE));
			}
			else if(MTTKRPmode == 3){
				checkCuda(hipMemcpy(dU + szDU[0] + szDU[1] , &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU + szDU[0] + szDU[1] + szDU[2], 0,  U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE));
			}
		}
		
		for (int m = 0; m < TiledX[0].ndims; ++m){

			/* matrix order according to mode order*/ 
			for (int mm = 0; mm < TiledX[0].ndims; ++mm){
				
				int curMode = TiledX[m].modeOrder[mm];
				dULoc[mm] = 0;
				
				for (int q = 0; q < curMode; ++q)
					dULoc[mm] +=  szDU[q % TiledX[0].ndims]; //1 2 3 0
			}	

			dBinLoc = 0;
			
			if(m > 0) {

				if (TiledX[m-1].totNnz > 0) {

					dLoc += TiledX[m-1].totNnz;
					dSlcLoc += TiledX[m - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[m - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[m - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[m - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[m - 1].fbrPtr[2].size(): 0) ;
				}
			}

			BLOCKSIZE = 512;
			dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

			if (TiledX[m].totNnz == 0) continue;

			cuda_timer_start(start);

			if(TiledX[m].modeOrder[0] == MTTKRPmode && TiledX[m].totNnz){

				if(Opt.verbose)
					cout << "Slc atomics - " ;

				// BLOCKSIZE = 128;
				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
				
				int warpPerFbr = Opt.warpPerSlice;//4;//;
				int logOfWarpPerFbr = log2(warpPerFbr);
				int fbrPerWarp = Opt.fiberPerWarp;//1;//BLOCKSIZE/32; // dont overflow TB
				int logOfFbrPerWarp = log2(fbrPerWarp );

				if( (warpPerFbr > (BLOCKSIZE/32)) || (fbrPerWarp > (BLOCKSIZE/32)) ){
					cout << "warpPerFbr (-w) or fbrPerWarp (-s) cannot be higher than threadblock size!"
					<< endl << "hint: increase -b!" << endl;
					exit(0);
				}

				grid.x = ( warpPerFbr * 32 * ((TiledX[m].nFibers + fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;
	
				if(TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, TiledX[m].nFibers, 
					dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
		
				else if(TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[m].nFibers, dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
			}

			else if(TiledX[0].ndims == 4 && TiledX[m].modeOrder[1] == MTTKRPmode && TiledX[m].totNnz){

				if(Opt.verbose)
					cout << "FbrS atomics - ";

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = Opt.warpPerSlice;//1;//BLOCKSIZE/32;//1;////4;//;	
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);

				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;
				
				mttkrp_MIHCSR_kernel_fbrS_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
				dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
				TiledX[m].nFibers, dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			}

			else if(TiledX[m].modeOrder[TiledX[0].ndims-2] == MTTKRPmode && TiledX[m].totNnz){
			
				if(Opt.verbose)
					cout << "Fbr atomics - ";

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				
				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				if(TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, TiledX[m].nFibers, 
					dU + dULoc[1], dU + dULoc[2], dU + dULoc[0], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				
				else if (TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[m].nFibers,  dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			}

			else if(TiledX[m].modeOrder[TiledX[0].ndims-1] == MTTKRPmode && TiledX[m].totNnz){

				if(Opt.verbose)
					cout << "nnz atomics - " ;

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				
				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				if (TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, TiledX[m].nFibers, 
					dU + dULoc[2], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr); 

				else if (TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[m].nFibers,  dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			}
		
			cuda_timer_stop(start, stop, mili);
		    GPUTime += mili;

		    if(Opt.verbose)
		    {
		    	cout << "Tile: " << m << " - time: " << mili << " ms";
		    	cout <<" nnz: " << TiledX[m].totNnz << " nFibers: "
		    	<< TiledX[m].fbrPtr[1].size() << " nSlc " << TiledX[m].fbrIdx[0].size() << " ";
				cout << " modeOrder: " << TiledX[m].modeOrder[0] <<" " << TiledX[m].modeOrder[1] <<" "
				<< TiledX[m].modeOrder[2];
				cout << endl;
			}   
		}
		if(Opt.verbose)
			cout << "MI-HCSR-GPU-mode "<< MTTKRPmode <<" : " << GPUTime << "," << endl;
		allModeGPUTime += GPUTime; 
	}
	int totalMIslics = 0, totalMISfibers = 0, totalMIfibers = 0, totalMInnz = 0;;
	for (int m = 0; m <  TiledX[0].ndims; ++m){
		if(TiledX[m].totNnz){
			if(TiledX[m].ndims == 3){
				totalMIslics += TiledX[m].fbrIdx[0].size();
				totalMIfibers += TiledX[m].fbrPtr[1].size();
				totalMInnz += TiledX[m].totNnz;
			}

			if(TiledX[m].ndims == 4){
				totalMIslics += TiledX[m].fbrIdx[0].size();
				totalMISfibers += TiledX[m].fbrPtr[1].size();
				totalMIfibers += TiledX[m].fbrPtr[2].size();
				totalMInnz += TiledX[m].totNnz;

			}
		}
	}

	cout << "Total GPU time: " << allModeGPUTime;
	
	// if(Opt.verbose)
	if(TiledX[0].ndims == 3)
		cout << " nSlc:" << totalMIslics
			<< ", nFibers:" << totalMIfibers << ", nnz:" << totalMInnz  
			<< endl;
	else if(TiledX[0].ndims == 4)
		cout << " nSlc:" << totalMIslics  << ", nSFibers:" << totalMISfibers
			<< ", nFibers:" << totalMIfibers << ", nnz:" << totalMInnz  
			<< endl;
	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);

	/* Copying output matrix from GPU to CPU for correctness check */
	int MTTKRPmode = TiledX[0].ndims - 1;
	ITYPE loc = ((TiledX[0].ndims == 3) ? szDU[0] + szDU[1] : szDU[0] + szDU[1] + szDU[2]);
	checkCuda(hipMemcpy(&U[MTTKRPmode].vals[0], dU + loc, U[MTTKRPmode].nRows * U[MTTKRPmode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	hipFree(dVals); 
	hipFree(dU); //hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int init_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt, ITYPE **dInds2, ITYPE **dfbrPtr1, ITYPE **dfbrIdx1, ITYPE **dFbrLikeSlcInds, DTYPE **dVals, DTYPE **dU){

	
	ITYPE mode0 = 0;//TiledX[0].modeOrder[0];
    ITYPE mode1 = 1;;//TiledX[0].modeOrder[1];
    ITYPE mode2 = 2;//TiledX[0].modeOrder[2];
    ITYPE mode3 = 3;//((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	// if(iter == 0 && cpdMode == 0)
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;
	
	for (int m = 0; m < TiledX[0].ndims; ++m){
		
		if (TiledX[m].totNnz == 0) continue;
		
		totNnz += TiledX[m].totNnz;
		totFbrPtr += TiledX[m].fbrPtr[1].size() ;
		totFbrIdx += TiledX[m].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[m].ndims == 4) ? TiledX[m].fbrPtr[2].size() : 0) ;
	}

	/*allocate and memcpy GPU memory*/
	checkCuda(hipMalloc((void**) dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) dInds2, totNnz * sizeof(ITYPE)), 0);
	

	for (int m = 0; m < TiledX[0].ndims; ++m){	

		if(m > 0) {

			if (TiledX[m-1].totNnz > 0) {
			
				dLoc += TiledX[m-1].totNnz;
				dSlcLoc += TiledX[m - 1].fbrPtr[0].size(); // all m same
				dSlcIdxLoc += TiledX[m - 1].fbrIdx[0].size(); 
				dFbrLoc += TiledX[m - 1].fbrPtr[1].size();
				dFbrIdxLoc += TiledX[m - 1].fbrIdx[1].size();
			}
		}

		if (TiledX[m].totNnz == 0) continue;

		checkCuda(hipMemcpy(*dVals + dLoc, &(TiledX[m].vals[0]), TiledX[m].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(*dfbrPtr1 + dFbrLoc, &(TiledX[m].fbrPtr[1][0]), TiledX[m].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(*dfbrIdx1 + dFbrIdxLoc, &(TiledX[m].fbrIdx[1][0]), TiledX[m].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(*dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[m].fbrLikeSlcInds[0]), TiledX[m].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(*dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
	}
	
	// //Matrices
    unsigned int *szDU =  new unsigned int[TiledX[0].ndims];

	ITYPE mtxSize = ((TiledX[0].ndims == 3) ? (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows) * U[mode0].nCols
		: (U[mode0].nRows + U[mode1].nRows + U[mode2].nRows + U[mode3].nRows) * U[mode0].nCols );
	
	for (int m = 0; m < TiledX[0].ndims; ++m)
		szDU[m] = U[m].nRows * U[m].nCols;

	checkCuda(hipMalloc((void**) dU, mtxSize * sizeof(DTYPE)), 0);
	
	// hipMemset(dU+0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(*dU + 0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(*dU + szDU[0], &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(*dU + szDU[0] + szDU[1], &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
    // MTTKRP_MIHCSR_GPU_oneMode_forCPD(TiledX, U, Opt, 0, 0,
 	//  dInds2, dfbrPtr1, dfbrIdx1, dFbrLikeSlcInds, dVals, dU);
}

int MTTKRP_MIHCSR_GPU_oneMode_forCPD(TiledTensor *TiledX, Matrix *U, const Options &Opt, int cpdMode, int iter,
	ITYPE *dInds2, ITYPE *dfbrPtr1, ITYPE *dfbrIdx1, ITYPE *dFbrLikeSlcInds, DTYPE *dVals, DTYPE *dU){

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili;
	
	ITYPE *dInds3, *dfbrPtr0, *dfbrIdx0, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;
	// DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	
	unsigned int *dULoc = new unsigned int[TiledX[0].ndims];
    unsigned int *szDU =  new unsigned int[TiledX[0].ndims];
	
	for (int m = 0; m < TiledX[0].ndims; ++m)
		szDU[m] = U[m].nRows * U[m].nCols;

	ITYPE loc = 0;
	for (int m = 0; m < cpdMode; ++m)
		loc += szDU[m];

	hipMemset(dU+loc, 0,  U[cpdMode].nRows * U[cpdMode].nCols * sizeof(DTYPE));
		
	// BLOCK and GRID
	int BLOCKSIZE = 512;
    float GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	int MTTKRPmode = cpdMode;
	// for (int MTTKRPmode = 0; MTTKRPmode < TiledX[0].ndims; ++MTTKRPmode)
	{	
		for (int m = 0; m < TiledX[0].ndims; ++m){

			/* matrix order according to mode order*/ 
			for (int mm = 0; mm < TiledX[0].ndims; ++mm){
				
				int curMode = TiledX[m].modeOrder[mm];
				dULoc[mm] = 0;
				
				for (int q = 0; q < curMode; ++q)
					dULoc[mm] +=  szDU[q % TiledX[0].ndims]; //1 2 3 0
			}	

			dBinLoc = 0;
			
			if(m > 0) {

				if (TiledX[m-1].totNnz > 0) {

					dLoc += TiledX[m-1].totNnz;
					dSlcLoc += TiledX[m - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[m - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[m - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[m - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[m - 1].fbrPtr[2].size(): 0) ;
				}
			}

			BLOCKSIZE = 512;
			dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

			if (TiledX[m].totNnz == 0) continue;

			cuda_timer_start(start);

			if(TiledX[m].modeOrder[0] == MTTKRPmode && TiledX[m].totNnz){

				// if(Opt.verbose)
					// cout << "Slc atomics - " ;

				// BLOCKSIZE = 128;
				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
				
				int warpPerFbr = Opt.warpPerSlice;//4;//;
				int logOfWarpPerFbr = log2(warpPerFbr);
				int fbrPerWarp = Opt.fiberPerWarp;//1;//BLOCKSIZE/32; // dont overflow TB
				int logOfFbrPerWarp = log2(fbrPerWarp );

				if( (warpPerFbr > (BLOCKSIZE/32)) || (fbrPerWarp > (BLOCKSIZE/32)) ){
					cout << "warpPerFbr (-w) or fbrPerWarp (-s) cannot be higher than threadblock size!"
					<< endl << "hint: increase -b!" << endl;
					exit(0);
				}

				grid.x = ( warpPerFbr * 32 * ((TiledX[m].nFibers + fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;
	
				if(TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, TiledX[m].nFibers, 
					dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
		
				else if(TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[m].nFibers, dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], dU + dULoc[3], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
			}

			else if(TiledX[m].modeOrder[TiledX[0].ndims-2] == MTTKRPmode && TiledX[m].totNnz){
			
				// if(Opt.verbose)
					// cout << "Fbr atomics - ";

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				
				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				if(TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, TiledX[m].nFibers, 
					dU + dULoc[1], dU + dULoc[2], dU + dULoc[0], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				
				else if (TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[m].nFibers,  dU + dULoc[2], dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			}

			else if(TiledX[m].modeOrder[TiledX[0].ndims-1] == MTTKRPmode && TiledX[m].totNnz){

				// if(Opt.verbose)
					// cout << "nnz atomics - " ;

				BLOCKSIZE = Opt.TBsize;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				if(warpPerFbr > (BLOCKSIZE/32)){
					cout << "warpPerFbr (-w) cannot be higher than threadblock size! hint: increase -b!" << endl;
					exit(0);
				}
				int logOfWarpPerFbr = log2(warpPerFbr);
				
				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

				if (TiledX[0].ndims == 3)
					mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, TiledX[m].nFibers, 
					dU + dULoc[2], dU + dULoc[0], dU + dULoc[1], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr); 

				else if (TiledX[0].ndims == 4)
					mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_4D<<<grid, block, 0, 0>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrIdxLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, 
					TiledX[m].nFibers,  dU + dULoc[3], dU + dULoc[0], dU + dULoc[1], dU + dULoc[2], Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			}
		
			cuda_timer_stop(start, stop, mili);
		    GPUTime += mili;

		    if(Opt.verbose)
		    {
		    	cout << "Tile: " << m << " - time: " << mili << " ms";
		    	cout <<" nnz: " << TiledX[m].totNnz << " nFibers: "
		    	<< TiledX[m].fbrPtr[1].size() << " nSlc " << TiledX[m].fbrIdx[0].size() << " ";
				cout << " modeOrder: " << TiledX[m].modeOrder[0] <<" " << TiledX[m].modeOrder[1] <<" "
				<< TiledX[m].modeOrder[2];
				cout << endl;
			}   
		}
		// cout << "MI-HCSR-GPU-mode "<< MTTKRPmode <<" : " << GPUTime << "," << endl;
		allModeGPUTime += GPUTime; 
	}
	// ITYPE loc = 0;
	// for (int m = 0; m < cpdMode; ++m)
	// 	loc += szDU[m];
	// ITYPE loc =  szDU[0];

	/* Copying output matrix from GPU to CPU for correctness check */
	checkCuda(hipMemcpy(&U[cpdMode].vals[0], dU + loc, U[cpdMode].nRows * U[cpdMode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	if(iter == Opt.cpdIters - 1 && cpdMode == TiledX[0].ndims - 1)
	{
		cout << "Freeing variable " << endl;
		hipFree(dVals); 
		hipFree(dU); //hipFree(dU1); hipFree(dU2); hipFree(dU3);
		hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
		hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
		hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
		hipFree(dFbrLikeSlcInds);
	}

	return 0;
}


